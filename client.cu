#include "hip/hip_runtime.h"
#include <arpa/inet.h>
#include <cstring>
#include <dlfcn.h>
#include <functional>
#include <iostream>
#include <netdb.h>
#include <nvml.h>
#include <pthread.h>
#include <stdio.h>
#include <string.h>
#include <string>
#include <sys/socket.h>
#include <sys/types.h>
#include <unistd.h>
#include <unordered_map>
#include <vector>

#include <unordered_map>

#include "api.h"

int sockfd = -1;
char *port;

int open_rpc_client()
{
    // if socket is already opened, return our socket.
    if (sockfd != -1)
    {
        // << "socket already opened" << std::endl;

        return sockfd;
    }

    // << "opening tcp socket..." << std::endl;

    char *server_ip = getenv("SCUDA_SERVER");
    if (server_ip == NULL)
    {
        printf("SCUDA_SERVER environment variable not set\n");
        std::exit(1);
    }

    char *p = getenv("SCUDA_PORT");

    if (p == NULL)
    {
        port = (char *)"14833";
    }
    else
    {
        port = p;
        // << "using SCUDA_PORT: " << port << std::endl;
    }

    addrinfo hints, *res;
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_STREAM;
    if (getaddrinfo(server_ip, port, &hints, &res) != 0)
    {
        printf("getaddrinfo failed\n");
        return -1;
    }

    sockfd = socket(res->ai_family, res->ai_socktype, res->ai_protocol);
    if (sockfd == -1)
    {
        printf("socket creation failed...\n");
        return -1;
    }

    if (connect(sockfd, res->ai_addr, res->ai_addrlen) != 0)
    {
        printf("connection with the server failed...\n");
        return -1;
    }
    return sockfd;
}

pthread_mutex_t mutex;
pthread_cond_t cond;

int rpc_start_request(const unsigned int op)
{
    static int next_request_id = 1;

    // write the request atomically
    pthread_mutex_lock(&mutex);

    int request_id = next_request_id++;

    if (write(sockfd, &request_id, sizeof(int)) < 0 ||
        write(sockfd, &op, sizeof(unsigned int)) < 0)
    {
        pthread_mutex_unlock(&mutex);
        return -1;
    }

    return request_id;
}

int rpc_write(const void *data, size_t size)
{
    if (write(sockfd, data, size) < 0)
    {
        pthread_mutex_unlock(&mutex);
        return -1;
    }
    return 0;
}

int rpc_read(void *data, size_t size)
{
    if (read(sockfd, data, size) < 0)
    {
        pthread_mutex_unlock(&mutex);
        return -1;
    }
    return 0;
}

int rpc_wait_for_response(int request_id)
{
    static int active_response_id = -1;

    // wait for the response
    while (true)
    {
        while (active_response_id != request_id && active_response_id != -1)
            pthread_cond_wait(&cond, &mutex);

        // we currently own mutex. if active response id is -1, read the response id
        if (active_response_id == -1)
        {
            if (read(sockfd, &active_response_id, sizeof(int)) < 0)
            {
                pthread_mutex_unlock(&mutex);
                return -1;
            }

            if (active_response_id != request_id)
            {
                pthread_cond_broadcast(&cond);
                continue;
            }
        }

        active_response_id = -1;
        return 0;
    }
}

nvmlReturn_t rpc_get_return(int request_id)
{
    nvmlReturn_t result;
    if (read(sockfd, &result, sizeof(nvmlReturn_t)) < 0)
        result = NVML_ERROR_GPU_IS_LOST;

    pthread_mutex_unlock(&mutex);
    return result;
}

void close_rpc_client()
{
    close(sockfd);
    sockfd = 0;
}

// 4.11 Initialization and Cleanup
nvmlReturn_t nvmlInitWithFlags(unsigned int flags)
{
    if (open_rpc_client() < 0)
        return NVML_ERROR_GPU_IS_LOST;
    int request_id = rpc_start_request(RPC_nvmlInitWithFlags);
    if (request_id < 0 ||
        rpc_write(&flags, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlInit_v2()
{
    if (open_rpc_client() < 0)
        return NVML_ERROR_GPU_IS_LOST;
    int request_id = rpc_start_request(RPC_nvmlInit_v2);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlShutdown()
{
    int request_id = rpc_start_request(RPC_nvmlShutdown);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    nvmlReturn_t result = rpc_get_return(request_id);
    close_rpc_client();
    return result;
}

// 4.14 System Queries
nvmlReturn_t nvmlSystemGetDriverVersion(char *version, unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetDriverVersion);
    if (request_id < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(version, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlSystemGetHicVersion(unsigned int *hwbcCount,
                                     nvmlHwbcEntry_t *hwbcEntries)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetHicVersion);
    if (request_id < 0 ||
        rpc_write(hwbcCount, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(hwbcCount, sizeof(unsigned int)) < 0 ||
        rpc_read(hwbcEntries, *hwbcCount * sizeof(nvmlHwbcEntry_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    nvmlReturn_t result = rpc_get_return(request_id);
    close_rpc_client();
    return result;
}

nvmlReturn_t nvmlSystemGetNVMLVersion(char *version, unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetNVMLVersion);
    if (request_id < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(version, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlSystemGetProcessName(unsigned int pid, char *name,
                                      unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetProcessName);
    if (request_id < 0 ||
        rpc_write(&pid, sizeof(unsigned int)) < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(name, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlSystemGetTopologyGpuSet(unsigned int cpuNumber,
                                         unsigned int *count,
                                         nvmlDevice_t *deviceArray)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetTopologyGpuSet);
    if (request_id < 0 ||
        rpc_write(&cpuNumber, sizeof(unsigned int)) < 0 ||
        rpc_write(count, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(count, sizeof(unsigned int)) < 0 ||
        rpc_read(deviceArray, *count * sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

// 4.15 Unit Queries
nvmlReturn_t nvmlUnitGetCount(unsigned int *unitCount)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetCount);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(unitCount, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlUnitGetDevices(nvmlUnit_t unit, unsigned int *deviceCount,
                                nvmlDevice_t *devices)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetDevices);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(deviceCount, sizeof(unsigned int)) < 0 ||
        rpc_read(devices, *deviceCount * sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlUnitGetFanSpeedInfo(nvmlUnit_t unit,
                                     nvmlUnitFanSpeeds_t *fanSpeeds)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetFanSpeedInfo);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(fanSpeeds, sizeof(nvmlUnitFanSpeeds_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlUnitGetHandleByIndex(unsigned int index, nvmlUnit_t *unit)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetHandleByIndex);
    if (request_id < 0 ||
        rpc_write(&index, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(unit, sizeof(nvmlUnit_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlUnitGetLedState(nvmlUnit_t unit, nvmlLedState_t *state)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetLedState);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(state, sizeof(nvmlLedState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlUnitGetPsuInfo(nvmlUnit_t unit, nvmlPSUInfo_t *psu)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetPsuInfo);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(psu, sizeof(nvmlPSUInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlUnitGetTemperature(nvmlUnit_t unit, unsigned int type,
                                    unsigned int *temp)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetTemperature);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_write(&type, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(temp, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlUnitGetUnitInfo(nvmlUnit_t unit, nvmlUnitInfo_t *info)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetUnitInfo);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(info, sizeof(nvmlUnitInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

// 4.16 Device Queries
nvmlReturn_t nvmlDeviceGetClkMonStatus(nvmlDevice_t device, nvmlClkMonStatus_t *status)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetClkMonStatus);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(status, sizeof(nvmlClkMonStatus_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetClock(nvmlDevice_t device, nvmlClockType_t clockType, nvmlClockId_t clockId, unsigned int *clockMHz)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetClock);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&clockType, sizeof(nvmlClockType_t)) < 0 ||
        rpc_write(&clockId, sizeof(nvmlClockId_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(clockMHz, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetClockInfo(nvmlDevice_t device, nvmlClockType_t type, unsigned int *clock)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetClockInfo);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&type, sizeof(nvmlClockType_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(clock, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetComputeMode(nvmlDevice_t device, nvmlComputeMode_t *mode)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetComputeMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(mode, sizeof(nvmlComputeMode_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetCount_v2(unsigned int *deviceCount)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetCount_v2);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(deviceCount, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetDisplayActive(nvmlDevice_t device, nvmlEnableState_t *isActive)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDisplayActive);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(isActive, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetDisplayMode(nvmlDevice_t device, nvmlEnableState_t *display)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDisplayMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(display, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetDriverModel_v2(nvmlDevice_t device, nvmlDriverModel_t *current, nvmlDriverModel_t *pending)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDriverModel_v2);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(current, sizeof(nvmlDriverModel_t)) < 0 ||
        rpc_read(pending, sizeof(nvmlDriverModel_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetDynamicPstatesInfo(nvmlDevice_t device, nvmlGpuDynamicPstatesInfo_t *pDynamicPstatesInfo)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDynamicPstatesInfo);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pDynamicPstatesInfo, sizeof(nvmlGpuDynamicPstatesInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetEccMode(nvmlDevice_t device, nvmlEnableState_t *current, nvmlEnableState_t *pending)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEccMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(current, sizeof(nvmlEnableState_t)) < 0 ||
        rpc_read(pending, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetEncoderCapacity(nvmlDevice_t device, nvmlEncoderType_t encoderQueryType, unsigned int *encoderCapacity)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderCapacity);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&encoderQueryType, sizeof(nvmlEncoderType_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(encoderCapacity, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetEncoderSessions(nvmlDevice_t device, unsigned int *sessionCount, nvmlEncoderSessionInfo_t *sessionInfos)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderSessions);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_read(sessionInfos, *sessionCount * sizeof(nvmlEncoderSessionInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetEncoderStats(nvmlDevice_t device, unsigned int *sessionCount, unsigned int *averageFps, unsigned int *averageLatency)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderStats);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_write(averageFps, sizeof(unsigned int)) < 0 ||
        rpc_write(averageLatency, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_read(averageFps, sizeof(unsigned int)) < 0 ||
        rpc_read(averageLatency, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetEncoderUtilization(nvmlDevice_t device, unsigned int *utilization, unsigned int *samplingPeriodUs)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderUtilization);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(utilization, sizeof(unsigned int)) < 0 ||
        rpc_write(samplingPeriodUs, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(utilization, sizeof(unsigned int)) < 0 ||
        rpc_read(samplingPeriodUs, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetEnforcedPowerLimit(nvmlDevice_t device, unsigned int *limit)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEnforcedPowerLimit);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(limit, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetFanSpeed(nvmlDevice_t device, unsigned int *speed)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetFanSpeed);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(speed, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetHandleByIndex_v2(unsigned int index,
                                           nvmlDevice_t *device)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetHandleByIndex_v2);
    if (request_id < 0 ||
        rpc_write(&index, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(device, sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetIndex(nvmlDevice_t device, unsigned int *index)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetIndex);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(index, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetMemoryInfo_v2(nvmlDevice_t device,
                                        nvmlMemory_v2_t *memoryInfo)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetMemoryInfo_v2);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(memoryInfo, sizeof(nvmlMemory_v2_t)) < 0)
    {
        std::cerr << "Failed to start RPC request" << std::endl;
        return NVML_ERROR_GPU_IS_LOST;
    }

    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetName(nvmlDevice_t device, char *name,
                               unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetName);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(name, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPciInfo_v3(nvmlDevice_t device, nvmlPciInfo_t *pci)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPciInfo_v3);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pci, sizeof(nvmlPciInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPcieSpeed(nvmlDevice_t device, unsigned int *pcieSpeed)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPcieSpeed);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(pcieSpeed, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pcieSpeed, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPcieThroughput(nvmlDevice_t device, nvmlPcieUtilCounter_t counter, unsigned int *value)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPcieThroughput);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&counter, sizeof(nvmlPcieUtilCounter_t)) < 0 ||
        rpc_write(value, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(value, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPerformanceState(nvmlDevice_t device, nvmlPstates_t *pState)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPerformanceState);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pState, sizeof(nvmlPstates_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPersistenceMode(nvmlDevice_t device,
                                          nvmlEnableState_t *mode)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPersistenceMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(mode, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPowerSource(nvmlDevice_t device, nvmlPowerSource_t *powerSource)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPowerSource);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(powerSource, sizeof(nvmlPowerSource_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPowerState(nvmlDevice_t device, nvmlPstates_t *pState)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPowerState);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pState, sizeof(nvmlPstates_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetPowerUsage(nvmlDevice_t device, unsigned int *power)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPowerUsage);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(power, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetProcessUtilization(nvmlDevice_t device, nvmlProcessUtilizationSample_t *utilization, unsigned int *processSamplesCount, unsigned long long lastSeenTimeStamp)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetProcessUtilization);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(processSamplesCount, sizeof(unsigned int)) < 0 ||
        rpc_write(&lastSeenTimeStamp, sizeof(unsigned long long)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(processSamplesCount, sizeof(unsigned int)) < 0 ||
        rpc_read(utilization, sizeof(nvmlProcessUtilizationSample_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetSupportedPerformanceStates(nvmlDevice_t device, nvmlPstates_t *pstates, unsigned int size)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetSupportedPerformanceStates);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(pstates, sizeof(nvmlPstates_t)) < 0 ||
        rpc_write(&size, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pstates, size * sizeof(nvmlPstates_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetTargetFanSpeed(nvmlDevice_t device, unsigned int fan, unsigned int *targetSpeed)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTargetFanSpeed);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&fan, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(targetSpeed, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetTemperature(nvmlDevice_t device, nvmlTemperatureSensors_t sensorType, unsigned int *temp)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTemperature);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&sensorType, sizeof(nvmlTemperatureSensors_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(temp, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetTemperatureThreshold(nvmlDevice_t device, nvmlTemperatureThresholds_t thresholdType, unsigned int *temp)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTemperatureThreshold);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&thresholdType, sizeof(nvmlTemperatureThresholds_t)) < 0 ||
        rpc_write(temp, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(temp, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetThermalSettings(nvmlDevice_t device, unsigned int sensorIndex, nvmlGpuThermalSettings_t *pThermalSettings)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetThermalSettings);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&sensorIndex, sizeof(unsigned int)) < 0 ||
        rpc_write(pThermalSettings, sizeof(nvmlGpuThermalSettings_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pThermalSettings, sizeof(nvmlGpuThermalSettings_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetTopologyCommonAncestor(nvmlDevice_t device1, nvmlDevice_t device2, nvmlGpuTopologyLevel_t *pathInfo)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTopologyCommonAncestor);
    if (request_id < 0 ||
        rpc_write(&device1, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&device2, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(pathInfo, sizeof(nvmlGpuTopologyLevel_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pathInfo, sizeof(nvmlGpuTopologyLevel_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetTopologyNearestGpus(nvmlDevice_t device, nvmlGpuTopologyLevel_t level, unsigned int *count, nvmlDevice_t *deviceArray)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTopologyNearestGpus);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&level, sizeof(nvmlGpuTopologyLevel_t)) < 0 ||
        rpc_write(count, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(count, sizeof(unsigned int)) < 0 ||
        rpc_read(deviceArray, *count * sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetTotalEnergyConsumption(nvmlDevice_t device, unsigned long long *energy)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTotalEnergyConsumption);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(energy, sizeof(unsigned long long)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetUUID(nvmlDevice_t device, char *uuid, unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetUUID);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(uuid, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceGetUtilizationRates(nvmlDevice_t device, nvmlUtilization_t *utilization)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetUtilizationRates);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(utilization, sizeof(nvmlUtilization_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceValidateInforom(nvmlDevice_t device)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceValidateInforom);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

// 4.17 Unit Commands
nvmlReturn_t nvmlUnitSetLedState(nvmlUnit_t unit, nvmlLedColor_t color)
{
    int request_id = rpc_start_request(RPC_nvmlUnitSetLedState);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_write(&color, sizeof(nvmlLedColor_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

// 4.20 Event Handling Methods
nvmlReturn_t nvmlDeviceGetSupportedEventTypes(nvmlDevice_t device, unsigned long long *eventTypes)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetSupportedEventTypes);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(eventTypes, sizeof(unsigned long long)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlDeviceRegisterEvents(nvmlDevice_t device, unsigned long long eventTypes, nvmlEventSet_t set)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceRegisterEvents);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&eventTypes, sizeof(unsigned long long)) < 0 ||
        rpc_write(&set, sizeof(nvmlEventSet_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlEventSetCreate(nvmlEventSet_t *set)
{
    int request_id = rpc_start_request(RPC_nvmlEventSetCreate);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(set, sizeof(nvmlEventSet_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlEventSetFree(nvmlEventSet_t set)
{
    int request_id = rpc_start_request(RPC_nvmlEventSetFree);
    if (request_id < 0 ||
        rpc_write(&set, sizeof(nvmlEventSet_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

nvmlReturn_t nvmlEventSetWait_v2(nvmlEventSet_t set, nvmlEventData_t *data, unsigned int timeoutms)
{
    int request_id = rpc_start_request(RPC_nvmlEventSetWait_v2);
    if (request_id < 0 ||
        rpc_write(&set, sizeof(nvmlEventSet_t)) < 0 ||
        rpc_write(&timeoutms, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(data, sizeof(nvmlEventData_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return(request_id);
}

std::unordered_map<std::string, void *> functionMap;

void initializeFunctionMap()
{
    // simple cache check to make sure we only init handlers on the first run
    if (functionMap.find("nvmlInit_v2") == functionMap.end())
    {
        // attach all handlers to our function map
        functionMap["nvmlInitWithFlags"] = (void *)nvmlInitWithFlags;
        functionMap["nvmlInit_v2"] = (void *)nvmlInit_v2;
        functionMap["nvmlShutdown"] = (void *)nvmlShutdown;
        functionMap["nvmlSystemGetDriverVersion"] =
            (void *)nvmlSystemGetDriverVersion;
        functionMap["nvmlSystemGetHicVersion"] = (void *)nvmlSystemGetHicVersion;
        functionMap["nvmlSystemGetNVMLVersion"] = (void *)nvmlSystemGetNVMLVersion;
        functionMap["nvmlSystemGetProcessName"] = (void *)nvmlSystemGetProcessName;
        functionMap["nvmlSystemGetTopologyGpuSet"] =
            (void *)nvmlSystemGetTopologyGpuSet;
        functionMap["nvmlUnitGetCount"] = (void *)nvmlUnitGetCount;
        functionMap["nvmlUnitGetDevices"] = (void *)nvmlUnitGetDevices;
        functionMap["nvmlUnitGetFanSpeedInfo"] = (void *)nvmlUnitGetFanSpeedInfo;
        functionMap["nvmlUnitGetHandleByIndex"] = (void *)nvmlUnitGetHandleByIndex;
        functionMap["nvmlUnitGetLedState"] = (void *)nvmlUnitGetLedState;
        functionMap["nvmlUnitGetPsuInfo"] = (void *)nvmlUnitGetPsuInfo;
        functionMap["nvmlUnitGetTemperature"] = (void *)nvmlUnitGetTemperature;
        functionMap["nvmlUnitGetUnitInfo"] = (void *)nvmlUnitGetUnitInfo;

        // 4.16 Device Queries
        functionMap["nvmlDeviceGetClkMonStatus"] = (void *)nvmlDeviceGetClkMonStatus;
        functionMap["nvmlDeviceGetClock"] = (void *)nvmlDeviceGetClock;
        functionMap["nvmlDeviceGetClockInfo"] = (void *)nvmlDeviceGetClockInfo;
        functionMap["nvmlDeviceGetComputeMode"] = (void *)nvmlDeviceGetComputeMode;
        functionMap["nvmlDeviceGetCount_v2"] = (void *)nvmlDeviceGetCount_v2;
        functionMap["nvmlDeviceGetDisplayActive"] = (void *)nvmlDeviceGetDisplayActive;
        functionMap["nvmlDeviceGetDisplayMode"] = (void *)nvmlDeviceGetDisplayMode;
        functionMap["nvmlDeviceGetDriverModel_v2"] = (void *)nvmlDeviceGetDriverModel_v2;
        functionMap["nvmlDeviceGetDynamicPstatesInfo"] = (void *)nvmlDeviceGetDynamicPstatesInfo;
        functionMap["nvmlDeviceGetEccMode"] = (void *)nvmlDeviceGetEccMode;
        functionMap["nvmlDeviceGetEncoderCapacity"] = (void *)nvmlDeviceGetEncoderCapacity;
        functionMap["nvmlDeviceGetEncoderSessions"] = (void *)nvmlDeviceGetEncoderSessions;
        functionMap["nvmlDeviceGetEncoderStats"] = (void *)nvmlDeviceGetEncoderStats;
        functionMap["nvmlDeviceGetEncoderUtilization"] = (void *)nvmlDeviceGetEncoderUtilization;
        functionMap["nvmlDeviceGetEnforcedPowerLimit"] = (void *)nvmlDeviceGetEnforcedPowerLimit;
        functionMap["nvmlDeviceGetFanSpeed"] = (void *)nvmlDeviceGetFanSpeed;
        functionMap["nvmlDeviceGetHandleByIndex_v2"] =
            (void *)nvmlDeviceGetHandleByIndex_v2;
        functionMap["nvmlDeviceGetIndex"] = (void *)nvmlDeviceGetIndex;
        functionMap["nvmlDeviceGetMemoryInfo_v2"] =
            (void *)nvmlDeviceGetMemoryInfo_v2;
        functionMap["nvmlDeviceGetName"] = (void *)nvmlDeviceGetName;
        functionMap["nvmlDeviceGetPciInfo_v3"] = (void *)nvmlDeviceGetPciInfo_v3;
        functionMap["nvmlDeviceGetPcieSpeed"] = (void *)nvmlDeviceGetPcieSpeed;
        functionMap["nvmlDeviceGetPcieThroughput"] = (void *)nvmlDeviceGetPcieThroughput;
        functionMap["nvmlDeviceGetPerformanceState"] = (void *)nvmlDeviceGetPerformanceState;
        functionMap["nvmlDeviceGetPersistenceMode"] =
            (void *)nvmlDeviceGetPersistenceMode;
        functionMap["nvmlDeviceGetPowerSource"] = (void *)nvmlDeviceGetPowerSource;
        functionMap["nvmlDeviceGetPowerState"] = (void *)nvmlDeviceGetPowerState;
        functionMap["nvmlDeviceGetPowerUsage"] = (void *)nvmlDeviceGetPowerUsage;
        functionMap["nvmlDeviceGetSupportedPerformanceStates"] =
            (void *)nvmlDeviceGetSupportedPerformanceStates;
        functionMap["nvmlDeviceGetTargetFanSpeed"] = (void *)nvmlDeviceGetTargetFanSpeed;
        functionMap["nvmlDeviceGetTemperature"] = (void *)nvmlDeviceGetTemperature;
        functionMap["nvmlDeviceGetTemperatureThreshold"] = (void *)nvmlDeviceGetTemperatureThreshold;
        functionMap["nvmlDeviceGetThermalSettings"] = (void *)nvmlDeviceGetThermalSettings;
        functionMap["nvmlDeviceGetTopologyCommonAncestor"] = (void *)nvmlDeviceGetTopologyCommonAncestor;
        functionMap["nvmlDeviceGetTopologyNearestGpus"] = (void *)nvmlDeviceGetTopologyNearestGpus;
        functionMap["nvmlDeviceGetTotalEnergyConsumption"] = (void *)nvmlDeviceGetTotalEnergyConsumption;
        functionMap["nvmlDeviceGetUUID"] = (void *)nvmlDeviceGetUUID;
        functionMap["nvmlDeviceGetUtilizationRates"] = (void *)nvmlDeviceGetUtilizationRates;
        functionMap["nvmlDeviceValidateInforom"] = (void *)nvmlDeviceValidateInforom;

        // 4.17 Unit Commands
        functionMap["nvmlUnitSetLedState"] = (void *)nvmlUnitSetLedState;

        // 4.20 Event Handling Methods
        functionMap["nvmlDeviceGetSupportedEventTypes"] = (void *)nvmlDeviceGetSupportedEventTypes;
        functionMap["nvmlDeviceRegisterEvents"] = (void *)nvmlDeviceRegisterEvents;
        functionMap["nvmlEventSetCreate"] = (void *)nvmlEventSetCreate;
        functionMap["nvmlEventSetFree"] = (void *)nvmlEventSetFree;
        functionMap["nvmlEventSetWait_v2"] = (void *)nvmlEventSetWait_v2;
    }
}

// Lookup function similar to dlsym
void *getFunctionByName(const char *name)
{
    auto it = functionMap.find(name);
    if (it != functionMap.end())
        return it->second;
    return nullptr;
}

void *dlsym(void *handle, const char *name) __THROW
{
    initializeFunctionMap();

    void *func = getFunctionByName(name);
    
    if (func != nullptr)
        return func;

    static void *(*real_dlsym)(void *, const char *) = NULL;
    if (real_dlsym == NULL)
    {
        // avoid calling dlsym recursively; use dlvsym to resolve dlsym itself
        real_dlsym = (void *(*)(void *, const char *))dlvsym(RTLD_NEXT, "dlsym",
                                                             "GLIBC_2.2.5");
    }

    if (!strcmp(name, "dlsym"))
        return (void *)dlsym;

    // if func symbol is not found in the handler mappings, return the real dlsym
    // resolution
    return real_dlsym(handle, name);
}
