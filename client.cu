#include "hip/hip_runtime.h"

#include <arpa/inet.h>
#include <cstring>
#include <dlfcn.h>
#include <functional>
#include <iostream>
#include <netdb.h>
#include <nvml.h>
#include <hip/hip_runtime.h>
#include <pthread.h>
#include <stdio.h>
#include <string.h>
#include <string>
#include <sys/socket.h>
#include <sys/types.h>
#include <unistd.h>
#include <unordered_map>
#include <vector>

#include <unordered_map>

#include "api.h"

int sockfd = -1;
char *port;

int open_rpc_client()
{
    // if socket is already opened, return our socket.
    if (sockfd != -1)
    {
        // << "socket already opened" << std::endl;

        return sockfd;
    }

    // << "opening tcp socket..." << std::endl;

    char *server_ip = getenv("SCUDA_SERVER");
    if (server_ip == NULL)
    {
        printf("SCUDA_SERVER environment variable not set\n");
        std::exit(1);
    }

    char *p = getenv("SCUDA_PORT");

    if (p == NULL)
    {
        port = (char *)"14833";
    }
    else
    {
        port = p;
        // << "using SCUDA_PORT: " << port << std::endl;
    }

    addrinfo hints, *res;
    memset(&hints, 0, sizeof(hints));
    hints.ai_family = AF_INET;
    hints.ai_socktype = SOCK_STREAM;
    if (getaddrinfo(server_ip, port, &hints, &res) != 0)
    {
        printf("getaddrinfo failed\n");
        return -1;
    }

    sockfd = socket(res->ai_family, res->ai_socktype, res->ai_protocol);
    if (sockfd == -1)
    {
        printf("socket creation failed...\n");
        return -1;
    }

    if (connect(sockfd, res->ai_addr, res->ai_addrlen) != 0)
    {
        printf("connection with the server failed...\n");
        return -1;
    }
    return sockfd;
}

pthread_mutex_t mutex;
pthread_cond_t cond;

int rpc_start_request(const unsigned int op) {
    static int next_request_id = 1; // Initialized once and retains value across function calls

    // Ensure socket is open
    if (sockfd < 0) {
        std::cerr << "Socket not open" << std::endl;
        return -1;
    }

    // Lock the mutex for atomic operation
    pthread_mutex_lock(&mutex);

    int request_id = next_request_id++; // Assign and then increment

    // Write the request ID and operation code
    if (write(sockfd, &request_id, sizeof(int)) < 0) {
        std::cerr << "Failed to write request_id. Error: " << strerror(errno) << std::endl;
        pthread_mutex_unlock(&mutex);
        return -1;
    }

    if (write(sockfd, &op, sizeof(unsigned int)) < 0) {
        std::cerr << "Failed to write operation code. Error: " << strerror(errno) << std::endl;
        pthread_mutex_unlock(&mutex);
        return -1;
    }

    pthread_mutex_unlock(&mutex);

    return request_id;
}

int rpc_write(const void *data, size_t size)
{
    if (write(sockfd, data, size) < 0)
    {
        pthread_mutex_unlock(&mutex);
        return -1;
    }
    return 0;
}

int rpc_read(void *data, size_t size)
{
    if (data == nullptr) {
        // temp buffer to discard data
        char tempBuffer[256];
        while (size > 0) {
            ssize_t bytesRead = read(sockfd, tempBuffer, std::min(size, sizeof(tempBuffer)));
            if (bytesRead < 0) {
                pthread_mutex_unlock(&mutex);
                return -1; // error if reading fails
            }
            size -= bytesRead;
        }
    } else if (read(sockfd, data, size) < 0)
    {
        pthread_mutex_unlock(&mutex);
        return -1;
    }
    return 0;
}

int rpc_wait_for_response(int request_id)
{
    static int active_response_id = -1;

    // wait for the response
    while (true)
    {
        while (active_response_id != request_id && active_response_id != -1)
            pthread_cond_wait(&cond, &mutex);

        // we currently own mutex. if active response id is -1, read the response id
        if (active_response_id == -1)
        {
            if (read(sockfd, &active_response_id, sizeof(int)) < 0)
            {
                pthread_mutex_unlock(&mutex);
                return -1;
            }

            if (active_response_id != request_id)
            {
                pthread_cond_broadcast(&cond);
                continue;
            }
        }

        active_response_id = -1;
        return 0;
    }
}

template <typename T>
T rpc_get_return(int request_id, T error_value)
{
    T result;
    if (read(sockfd, &result, sizeof(T)) < 0)
        result = error_value;

    pthread_mutex_unlock(&mutex);

    return result;
}

void close_rpc_client()
{
    close(sockfd);
    sockfd = 0;
}

// 4.11 Initialization and Cleanup
nvmlReturn_t nvmlInitWithFlags(unsigned int flags)
{
    if (open_rpc_client() < 0)
        return NVML_ERROR_GPU_IS_LOST;
    int request_id = rpc_start_request(RPC_nvmlInitWithFlags);
    if (request_id < 0 ||
        rpc_write(&flags, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;

    nvmlReturn_t result = rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
    return result;
}

nvmlReturn_t nvmlInit_v2()
{
    if (open_rpc_client() < 0)
        return NVML_ERROR_GPU_IS_LOST;
    int request_id = rpc_start_request(RPC_nvmlInit_v2);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlShutdown()
{
    int request_id = rpc_start_request(RPC_nvmlShutdown);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    nvmlReturn_t result = rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
    close_rpc_client();
    return result;
}

// 4.14 System Queries
nvmlReturn_t nvmlSystemGetDriverVersion(char *version, unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetDriverVersion);
    if (request_id < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(version, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlSystemGetHicVersion(unsigned int *hwbcCount,
                                     nvmlHwbcEntry_t *hwbcEntries)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetHicVersion);
    if (request_id < 0 ||
        rpc_write(hwbcCount, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(hwbcCount, sizeof(unsigned int)) < 0 ||
        rpc_read(hwbcEntries, *hwbcCount * sizeof(nvmlHwbcEntry_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    nvmlReturn_t result = rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
    close_rpc_client();
    return result;
}

nvmlReturn_t nvmlSystemGetNVMLVersion(char *version, unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetNVMLVersion);
    if (request_id < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(version, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlSystemGetProcessName(unsigned int pid, char *name,
                                      unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetProcessName);
    if (request_id < 0 ||
        rpc_write(&pid, sizeof(unsigned int)) < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(name, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlSystemGetTopologyGpuSet(unsigned int cpuNumber,
                                         unsigned int *count,
                                         nvmlDevice_t *deviceArray)
{
    int request_id = rpc_start_request(RPC_nvmlSystemGetTopologyGpuSet);
    if (request_id < 0 ||
        rpc_write(&cpuNumber, sizeof(unsigned int)) < 0 ||
        rpc_write(count, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(count, sizeof(unsigned int)) < 0 ||
        rpc_read(deviceArray, *count * sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

// 4.15 Unit Queries
nvmlReturn_t nvmlUnitGetCount(unsigned int *unitCount)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetCount);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(unitCount, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlUnitGetDevices(nvmlUnit_t unit, unsigned int *deviceCount,
                                nvmlDevice_t *devices)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetDevices);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(deviceCount, sizeof(unsigned int)) < 0 ||
        rpc_read(devices, *deviceCount * sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlUnitGetFanSpeedInfo(nvmlUnit_t unit,
                                     nvmlUnitFanSpeeds_t *fanSpeeds)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetFanSpeedInfo);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(fanSpeeds, sizeof(nvmlUnitFanSpeeds_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlUnitGetHandleByIndex(unsigned int index, nvmlUnit_t *unit)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetHandleByIndex);
    if (request_id < 0 ||
        rpc_write(&index, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(unit, sizeof(nvmlUnit_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlUnitGetLedState(nvmlUnit_t unit, nvmlLedState_t *state)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetLedState);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(state, sizeof(nvmlLedState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlUnitGetPsuInfo(nvmlUnit_t unit, nvmlPSUInfo_t *psu)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetPsuInfo);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(psu, sizeof(nvmlPSUInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlUnitGetTemperature(nvmlUnit_t unit, unsigned int type,
                                    unsigned int *temp)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetTemperature);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_write(&type, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(temp, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlUnitGetUnitInfo(nvmlUnit_t unit, nvmlUnitInfo_t *info)
{
    int request_id = rpc_start_request(RPC_nvmlUnitGetUnitInfo);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(info, sizeof(nvmlUnitInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

// 4.16 Device Queries
nvmlReturn_t nvmlDeviceGetClkMonStatus(nvmlDevice_t device, nvmlClkMonStatus_t *status)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetClkMonStatus);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(status, sizeof(nvmlClkMonStatus_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetClock(nvmlDevice_t device, nvmlClockType_t clockType, nvmlClockId_t clockId, unsigned int *clockMHz)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetClock);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&clockType, sizeof(nvmlClockType_t)) < 0 ||
        rpc_write(&clockId, sizeof(nvmlClockId_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(clockMHz, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetClockInfo(nvmlDevice_t device, nvmlClockType_t type, unsigned int *clock)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetClockInfo);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&type, sizeof(nvmlClockType_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(clock, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetComputeMode(nvmlDevice_t device, nvmlComputeMode_t *mode)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetComputeMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(mode, sizeof(nvmlComputeMode_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetCount_v2(unsigned int *deviceCount)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetCount_v2);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(deviceCount, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetDisplayActive(nvmlDevice_t device, nvmlEnableState_t *isActive)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDisplayActive);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(isActive, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetDisplayMode(nvmlDevice_t device, nvmlEnableState_t *display)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDisplayMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(display, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetDriverModel_v2(nvmlDevice_t device, nvmlDriverModel_t *current, nvmlDriverModel_t *pending)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDriverModel_v2);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(current, sizeof(nvmlDriverModel_t)) < 0 ||
        rpc_read(pending, sizeof(nvmlDriverModel_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetDynamicPstatesInfo(nvmlDevice_t device, nvmlGpuDynamicPstatesInfo_t *pDynamicPstatesInfo)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetDynamicPstatesInfo);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pDynamicPstatesInfo, sizeof(nvmlGpuDynamicPstatesInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetEccMode(nvmlDevice_t device, nvmlEnableState_t *current, nvmlEnableState_t *pending)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEccMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(current, sizeof(nvmlEnableState_t)) < 0 ||
        rpc_read(pending, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetEncoderCapacity(nvmlDevice_t device, nvmlEncoderType_t encoderQueryType, unsigned int *encoderCapacity)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderCapacity);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&encoderQueryType, sizeof(nvmlEncoderType_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(encoderCapacity, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetEncoderSessions(nvmlDevice_t device, unsigned int *sessionCount, nvmlEncoderSessionInfo_t *sessionInfos)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderSessions);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_read(sessionInfos, *sessionCount * sizeof(nvmlEncoderSessionInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetEncoderStats(nvmlDevice_t device, unsigned int *sessionCount, unsigned int *averageFps, unsigned int *averageLatency)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderStats);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_write(averageFps, sizeof(unsigned int)) < 0 ||
        rpc_write(averageLatency, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(sessionCount, sizeof(unsigned int)) < 0 ||
        rpc_read(averageFps, sizeof(unsigned int)) < 0 ||
        rpc_read(averageLatency, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetEncoderUtilization(nvmlDevice_t device, unsigned int *utilization, unsigned int *samplingPeriodUs)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEncoderUtilization);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(utilization, sizeof(unsigned int)) < 0 ||
        rpc_write(samplingPeriodUs, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(utilization, sizeof(unsigned int)) < 0 ||
        rpc_read(samplingPeriodUs, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetEnforcedPowerLimit(nvmlDevice_t device, unsigned int *limit)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetEnforcedPowerLimit);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(limit, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetFanSpeed(nvmlDevice_t device, unsigned int *speed)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetFanSpeed);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(speed, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetHandleByIndex_v2(unsigned int index,
                                           nvmlDevice_t *device)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetHandleByIndex_v2);
    if (request_id < 0 ||
        rpc_write(&index, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(device, sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetIndex(nvmlDevice_t device, unsigned int *index)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetIndex);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(index, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetMemoryInfo_v2(nvmlDevice_t device,
                                        nvmlMemory_v2_t *memoryInfo)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetMemoryInfo_v2);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(memoryInfo, sizeof(nvmlMemory_v2_t)) < 0)
    {
        
        return NVML_ERROR_GPU_IS_LOST;
    }

    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetName(nvmlDevice_t device, char *name,
                               unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetName);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 || rpc_read(name, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPciInfo_v3(nvmlDevice_t device, nvmlPciInfo_t *pci)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPciInfo_v3);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pci, sizeof(nvmlPciInfo_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPcieSpeed(nvmlDevice_t device, unsigned int *pcieSpeed)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPcieSpeed);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(pcieSpeed, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pcieSpeed, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPcieThroughput(nvmlDevice_t device, nvmlPcieUtilCounter_t counter, unsigned int *value)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPcieThroughput);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&counter, sizeof(nvmlPcieUtilCounter_t)) < 0 ||
        rpc_write(value, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(value, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPerformanceState(nvmlDevice_t device, nvmlPstates_t *pState)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPerformanceState);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pState, sizeof(nvmlPstates_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPersistenceMode(nvmlDevice_t device,
                                          nvmlEnableState_t *mode)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPersistenceMode);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(mode, sizeof(nvmlEnableState_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPowerSource(nvmlDevice_t device, nvmlPowerSource_t *powerSource)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPowerSource);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(powerSource, sizeof(nvmlPowerSource_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPowerState(nvmlDevice_t device, nvmlPstates_t *pState)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPowerState);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pState, sizeof(nvmlPstates_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetPowerUsage(nvmlDevice_t device, unsigned int *power)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetPowerUsage);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(power, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetProcessUtilization(nvmlDevice_t device, nvmlProcessUtilizationSample_t *utilization, unsigned int *processSamplesCount, unsigned long long lastSeenTimeStamp)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetProcessUtilization);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(processSamplesCount, sizeof(unsigned int)) < 0 ||
        rpc_write(&lastSeenTimeStamp, sizeof(unsigned long long)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(processSamplesCount, sizeof(unsigned int)) < 0 ||
        rpc_read(utilization, sizeof(nvmlProcessUtilizationSample_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetSupportedPerformanceStates(nvmlDevice_t device, nvmlPstates_t *pstates, unsigned int size)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetSupportedPerformanceStates);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(pstates, sizeof(nvmlPstates_t)) < 0 ||
        rpc_write(&size, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pstates, size * sizeof(nvmlPstates_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetTargetFanSpeed(nvmlDevice_t device, unsigned int fan, unsigned int *targetSpeed)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTargetFanSpeed);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&fan, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(targetSpeed, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetTemperature(nvmlDevice_t device, nvmlTemperatureSensors_t sensorType, unsigned int *temp)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTemperature);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&sensorType, sizeof(nvmlTemperatureSensors_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(temp, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetTemperatureThreshold(nvmlDevice_t device, nvmlTemperatureThresholds_t thresholdType, unsigned int *temp)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTemperatureThreshold);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&thresholdType, sizeof(nvmlTemperatureThresholds_t)) < 0 ||
        rpc_write(temp, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(temp, sizeof(unsigned int)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetThermalSettings(nvmlDevice_t device, unsigned int sensorIndex, nvmlGpuThermalSettings_t *pThermalSettings)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetThermalSettings);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&sensorIndex, sizeof(unsigned int)) < 0 ||
        rpc_write(pThermalSettings, sizeof(nvmlGpuThermalSettings_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pThermalSettings, sizeof(nvmlGpuThermalSettings_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetTopologyCommonAncestor(nvmlDevice_t device1, nvmlDevice_t device2, nvmlGpuTopologyLevel_t *pathInfo)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTopologyCommonAncestor);
    if (request_id < 0 ||
        rpc_write(&device1, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&device2, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(pathInfo, sizeof(nvmlGpuTopologyLevel_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pathInfo, sizeof(nvmlGpuTopologyLevel_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetTopologyNearestGpus(nvmlDevice_t device, nvmlGpuTopologyLevel_t level, unsigned int *count, nvmlDevice_t *deviceArray)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTopologyNearestGpus);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&level, sizeof(nvmlGpuTopologyLevel_t)) < 0 ||
        rpc_write(count, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(count, sizeof(unsigned int)) < 0 ||
        rpc_read(deviceArray, *count * sizeof(nvmlDevice_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetTotalEnergyConsumption(nvmlDevice_t device, unsigned long long *energy)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetTotalEnergyConsumption);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(energy, sizeof(unsigned long long)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetUUID(nvmlDevice_t device, char *uuid, unsigned int length)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetUUID);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&length, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(uuid, length) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceGetUtilizationRates(nvmlDevice_t device, nvmlUtilization_t *utilization)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetUtilizationRates);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(utilization, sizeof(nvmlUtilization_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceValidateInforom(nvmlDevice_t device)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceValidateInforom);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

// 4.17 Unit Commands
nvmlReturn_t nvmlUnitSetLedState(nvmlUnit_t unit, nvmlLedColor_t color)
{
    int request_id = rpc_start_request(RPC_nvmlUnitSetLedState);
    if (request_id < 0 ||
        rpc_write(&unit, sizeof(nvmlUnit_t)) < 0 ||
        rpc_write(&color, sizeof(nvmlLedColor_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

// 4.20 Event Handling Methods
nvmlReturn_t nvmlDeviceGetSupportedEventTypes(nvmlDevice_t device, unsigned long long *eventTypes)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceGetSupportedEventTypes);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(eventTypes, sizeof(unsigned long long)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlDeviceRegisterEvents(nvmlDevice_t device, unsigned long long eventTypes, nvmlEventSet_t set)
{
    int request_id = rpc_start_request(RPC_nvmlDeviceRegisterEvents);
    if (request_id < 0 ||
        rpc_write(&device, sizeof(nvmlDevice_t)) < 0 ||
        rpc_write(&eventTypes, sizeof(unsigned long long)) < 0 ||
        rpc_write(&set, sizeof(nvmlEventSet_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlEventSetCreate(nvmlEventSet_t *set)
{
    int request_id = rpc_start_request(RPC_nvmlEventSetCreate);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(set, sizeof(nvmlEventSet_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlEventSetFree(nvmlEventSet_t set)
{
    int request_id = rpc_start_request(RPC_nvmlEventSetFree);
    if (request_id < 0 ||
        rpc_write(&set, sizeof(nvmlEventSet_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

nvmlReturn_t nvmlEventSetWait_v2(nvmlEventSet_t set, nvmlEventData_t *data, unsigned int timeoutms)
{
    int request_id = rpc_start_request(RPC_nvmlEventSetWait_v2);
    if (request_id < 0 ||
        rpc_write(&set, sizeof(nvmlEventSet_t)) < 0 ||
        rpc_write(&timeoutms, sizeof(unsigned int)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(data, sizeof(nvmlEventData_t)) < 0)
        return NVML_ERROR_GPU_IS_LOST;
    return rpc_get_return<nvmlReturn_t>(request_id, NVML_ERROR_GPU_IS_LOST);
}

hipError_t hiprtcLinkCreate(unsigned int numOptions, hipJitOption *options, void **optionValues, hiprtcLinkState *stateOut)
{
    std::cout << "calling hiprtcLinkCreate " << std::endl;
}

hipError_t hiprtcLinkAddData(hiprtcLinkState state, hipJitInputType type, void *data, size_t size, const char *name, unsigned int numOptions, hipJitOption *options, void **optionValues)
{
    
    int request_id = rpc_start_request(RPC_cuLinkAddData_v2);
    if (request_id < 0 ||
        rpc_write(&state, sizeof(hiprtcLinkState)) < 0 ||
        rpc_write(&type, sizeof(hipJitInputType)) < 0 ||
        rpc_write(&size, sizeof(size_t)) < 0 ||
        rpc_write(name, strlen(name) + 1) < 0 ||
        rpc_write(data, size) < 0 ||
        rpc_write(&numOptions, sizeof(unsigned int)) < 0 ||
        rpc_write(options, numOptions * sizeof(hipJitOption)) < 0 ||
        rpc_write(optionValues, numOptions * sizeof(void *)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hiprtcLinkComplete(hiprtcLinkState state, void **cubinOut, size_t *sizeOut)
{
    
    int request_id = rpc_start_request(RPC_cuLinkComplete);
    if (request_id < 0 ||
        rpc_write(&state, sizeof(hiprtcLinkState)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(cubinOut, sizeof(void *)) < 0 ||
        rpc_read(sizeOut, sizeof(size_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipModuleLoadData(hipModule_t *module, const void *image)
{
    
    int request_id = rpc_start_request(RPC_cuModuleLoadData);
    if (request_id < 0 ||
        rpc_write(image, sizeof(image)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(module, sizeof(hipModule_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipModuleUnload(hipModule_t hmod)
{
    
    int request_id = rpc_start_request(RPC_cuModuleUnload);
    if (request_id < 0 ||
        rpc_write(&hmod, sizeof(hipModule_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipDrvGetErrorString(hipError_t error, const char **pStr)
{
    
    int request_id = rpc_start_request(RPC_cuGetErrorString);
    if (request_id < 0 ||
        rpc_write(&error, sizeof(hipError_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pStr, sizeof(const char *)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hiprtcLinkDestroy(hiprtcLinkState state)
{
    
    int request_id = rpc_start_request(RPC_cuLinkDestroy);
    if (request_id < 0 ||
        rpc_write(&state, sizeof(hiprtcLinkState)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipModuleGetFunction(hipFunction_t *hfunc, hipModule_t hmod, const char *name)
{
    
    int request_id = rpc_start_request(RPC_cuModuleGetFunction);
    if (request_id < 0 ||
        rpc_write(&hmod, sizeof(hipModule_t)) < 0 ||
        rpc_write(name, strlen(name) + 1) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(hfunc, sizeof(hipFunction_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t cuFuncSetAttribute(hipFunction_t hfunc, hipFunction_attribute attrib, int value)
{
    
    int request_id = rpc_start_request(RPC_cuFuncSetAttribute);
    if (request_id < 0 ||
        rpc_write(&hfunc, sizeof(hipFunction_t)) < 0 ||
        rpc_write(&attrib, sizeof(hipFunction_attribute)) < 0 ||
        rpc_write(&value, sizeof(int)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipModuleLaunchKernel(hipFunction_t f, unsigned int gridDimX, unsigned int gridDimY, unsigned int gridDimZ, unsigned int blockDimX, unsigned int blockDimY, unsigned int blockDimZ, unsigned int sharedMemBytes, hipStream_t hStream, void **kernelParams, void **extra)
{
    
    int request_id = rpc_start_request(RPC_cuLaunchKernel);
    if (request_id < 0 ||
        rpc_write(&f, sizeof(hipFunction_t)) < 0 ||
        rpc_write(&gridDimX, sizeof(unsigned int)) < 0 ||
        rpc_write(&gridDimY, sizeof(unsigned int)) < 0 ||
        rpc_write(&gridDimZ, sizeof(unsigned int)) < 0 ||
        rpc_write(&blockDimX, sizeof(unsigned int)) < 0 ||
        rpc_write(&blockDimY, sizeof(unsigned int)) < 0 ||
        rpc_write(&blockDimZ, sizeof(unsigned int)) < 0 ||
        rpc_write(&sharedMemBytes, sizeof(unsigned int)) < 0 ||
        rpc_write(&hStream, sizeof(hipStream_t)) < 0 ||
        rpc_write(kernelParams, sizeof(void *)) < 0 ||
        rpc_write(extra, sizeof(void *)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipDrvGetErrorName(hipError_t error, const char **pStr)
{
    
    int request_id = rpc_start_request(RPC_cuGetErrorName);
    if (request_id < 0 ||
        rpc_write(&error, sizeof(hipError_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pStr, sizeof(const char *)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipModuleLoadFatBinary(hipModule_t *module, const void *fatCubin)
{
    
    int request_id = rpc_start_request(RPC_cuModuleLoadFatBinary);
    if (request_id < 0 ||
        rpc_write(fatCubin, sizeof(fatCubin)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(module, sizeof(hipModule_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipModuleLoadDataEx(hipModule_t *module, const void *image, unsigned int numOptions, hipJitOption *options, void **optionValues)
{
    
    int request_id = rpc_start_request(RPC_cuModuleLoadDataEx);
    if (request_id < 0 ||
        rpc_write(image, sizeof(image)) < 0 ||
        rpc_write(&numOptions, sizeof(unsigned int)) < 0 ||
        rpc_write(options, numOptions * sizeof(hipJitOption)) < 0 ||
        rpc_write(optionValues, numOptions * sizeof(void *)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(module, sizeof(hipModule_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hiprtcLinkAddFile(hiprtcLinkState state, hipJitInputType type, const char *path, unsigned int numOptions, hipJitOption *options, void **optionValues)
{
    
    int request_id = rpc_start_request(RPC_cuLinkAddFile_v2);
    if (request_id < 0 ||
        rpc_write(&state, sizeof(hiprtcLinkState)) < 0 ||
        rpc_write(&type, sizeof(hipJitInputType)) < 0 ||
        rpc_write(path, strlen(path) + 1) < 0 ||
        rpc_write(&numOptions, sizeof(unsigned int)) < 0 ||
        rpc_write(options, numOptions * sizeof(hipJitOption)) < 0 ||
        rpc_write(optionValues, numOptions * sizeof(void *)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipInit(unsigned int flags) {
    std::cerr << "calling cuinit1!!!" << std::endl; 

    // Open RPC client if not already opened
    if (open_rpc_client() < 0)
        return hipErrorUnknown;

    // Start the RPC request for hipInit
    int request_id = rpc_start_request(RPC_cuInit);
    if (request_id < 0) {
        std::cerr << "Failed to start hipInit request" << std::endl;
        return hipErrorUnknown;
    }

    // Write the flags to the server
    if (rpc_write(&flags, sizeof(unsigned int)) < 0) {
        std::cerr << "Failed to write flags to server" << std::endl;
        return hipErrorUnknown;
    }

    // Wait for the server response
    if (rpc_wait_for_response(request_id) < 0) {
        std::cerr << "Failed to wait for response from server" << std::endl;
        return hipErrorUnknown;
    }

    // Read the result code from the server
    hipError_t result;
    if (rpc_read(&result, sizeof(hipError_t)) < 0) {
        std::cerr << "Failed to read result code from server" << std::endl;
        return hipErrorUnknown;
    }

    // Log the successful initialization
    if (result == hipSuccess) {
        std::cout << "hipInit successful, Flags: " << flags << std::endl;
    }

    // Return the result received from the server
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipCtxPushCurrent(hipCtx_t ctx)
{
    
    int request_id = rpc_start_request(RPC_cuCtxPushCurrent);
    if (request_id < 0 ||
        rpc_write(&ctx, sizeof(hipCtx_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipCtxPopCurrent(hipCtx_t *pctx)
{
    
    int request_id = rpc_start_request(RPC_cuCtxPopCurrent);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pctx, sizeof(hipCtx_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipCtxGetDevice(hipDevice_t *device)
{
    
    int request_id = rpc_start_request(RPC_cuCtxGetDevice);
    if (request_id < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(device, sizeof(hipDevice_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipDevicePrimaryCtxRetain(hipCtx_t *pctx, hipDevice_t dev)
{
    
    int request_id = rpc_start_request(RPC_cuDevicePrimaryCtxRetain);
    if (request_id < 0 ||
        rpc_write(&dev, sizeof(hipDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pctx, sizeof(hipCtx_t)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipDevicePrimaryCtxRelease(hipDevice_t dev)
{
    
    int request_id = rpc_start_request(RPC_cuDevicePrimaryCtxRelease);
    if (request_id < 0 ||
        rpc_write(&dev, sizeof(hipDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipDevicePrimaryCtxReset(hipDevice_t dev)
{
    
    int request_id = rpc_start_request(RPC_cuDevicePrimaryCtxReset);
    if (request_id < 0 ||
        rpc_write(&dev, sizeof(hipDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipDeviceGetAttribute(int *pi, hipDeviceAttribute_t attrib, hipDevice_t dev)
{
    
    int request_id = rpc_start_request(RPC_cuDeviceGetAttribute);
    if (request_id < 0 ||
        rpc_write(&attrib, sizeof(hipDeviceAttribute_t)) < 0 ||
        rpc_write(&dev, sizeof(hipDevice_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(pi, sizeof(int)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipStreamSynchronize(hipStream_t hStream)
{
    
    int request_id = rpc_start_request(RPC_cuStreamSynchronize);
    if (request_id < 0 ||
        rpc_write(&hStream, sizeof(hipStream_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t cuDeviceGetShim(hipDevice_t *device, int ordinal) {
    std::cout << "Client: calling cuDeviceGetShim" << std::endl;

    // Check if the device pointer is valid
    if (device == nullptr) {
        std::cerr << "Invalid device pointer provided." << std::endl;
        return hipErrorInvalidValue;
    }

    // Ensure the socket is connected
    if (sockfd < 0) {
        std::cerr << "Socket not connected." << std::endl;
        return hipErrorUnknown;
    }

    // Start the request with the specific RPC operation code for hipDeviceGet
    int request_id = rpc_start_request(RPC_cuDeviceGet);
    if (request_id < 0) {
        std::cerr << "Failed to start request for hipDeviceGet" << std::endl;
        return hipErrorUnknown;
    }

    // Write the ordinal value to the server
    if (rpc_write(&ordinal, sizeof(int)) < 0) {
        std::cerr << "Failed to write ordinal to server. Error: " << strerror(errno) << std::endl;
        return hipErrorUnknown;
    }

    // Wait for the server's response
    if (rpc_wait_for_response(request_id) < 0) {
        std::cerr << "Failed to wait for response from server" << std::endl;
        return hipErrorUnknown;
    }

    // Read the result code from the server
    hipError_t result;
    if (rpc_read(&result, sizeof(hipError_t)) < 0) {
        std::cerr << "Failed to read result code from server. Error: " << strerror(errno) << std::endl;
        return hipErrorUnknown;
    }

    // Check if the hipDeviceGet call was successful
    if (result != hipSuccess) {
        std::cerr << "hipDeviceGet call failed on the server. Error code: " << result << std::endl;
        return result;
    }

    // Read the device handle from the server
    if (rpc_read(device, sizeof(hipDevice_t)) < 0) {
        std::cerr << "Failed to read device handle from server. Error: " << strerror(errno) << std::endl;
        return hipErrorUnknown;
    }

    std::cout << "Client: Received device handle from server: " << *device << std::endl;

    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t cuDeviceGetCountShim(int *deviceCount) {
    std::cout << "Client: calling cuDeviceGetCountShim" << std::endl;

    if (sockfd < 0) {
        std::cerr << "Socket not connected." << std::endl;
        return hipErrorUnknown;
    }

    int request_id = rpc_start_request(RPC_cuDeviceGetCount);
    if (request_id < 0) {
        std::cerr << "Failed to start request for hipGetDeviceCount" << std::endl;
        return hipErrorUnknown;
    }

    if (rpc_wait_for_response(request_id) < 0) {
        std::cerr << "Failed to wait for response from server" << std::endl;
        return hipErrorUnknown;
    }

    hipError_t result;
    ssize_t bytes_read = rpc_read(&result, sizeof(hipError_t));
    if (bytes_read < 0) {
        std::cerr << "Failed to read result from server. Error: " << strerror(errno) << std::endl;
        return hipErrorUnknown;
    }

    if (result != hipSuccess) {
        std::cerr << "hipGetDeviceCount call failed on the server. Error code: " << result << std::endl;
        return result;
    }

    bytes_read = rpc_read(deviceCount, sizeof(int));
    if (bytes_read < 0) {
        std::cerr << "Failed to read device count from server. Error: " << strerror(errno) << std::endl;
        return hipErrorUnknown;
    }

    std::cout << "Client: Received device count from server: " << *deviceCount << std::endl;

    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

void cuDeviceGetNameShim() {
    std::cout << "calling cuDeviceGetNameShim" << std::endl;
}

void cuDeviceTotalMemShim() {
    std::cout << "calling cuDeviceTotalMemShim" << std::endl;
}

void cuDeviceGetAttributeShim() {
    std::cout << "calling cuDeviceGetAttributeShim" << std::endl;
}

void cuDeviceGetP2PAttributeShim() {
    std::cout << "calling cuDeviceGetP2PAttributeShim" << std::endl;
}

void cuDeviceGetByPCIBusIdShim() {
    std::cout << "calling cuDeviceGetByPCIBusIdShim" << std::endl;
}

void cuDeviceGetPCIBusIdShim() {
    std::cout << "calling cuDeviceGetPCIBusIdShim" << std::endl;
}

void cuDeviceGetUuidShim() {
    std::cout << "calling cuDeviceGetUuidShim" << std::endl;
}

void cuDeviceGetTexture1DLinearMaxWidthShim() {
    std::cout << "calling cuDeviceGetTexture1DLinearMaxWidthShim" << std::endl;
}

void cuDeviceGetDefaultMemPoolShim() {
    std::cout << "calling cuDeviceGetDefaultMemPoolShim" << std::endl;
}

void cuDeviceSetMemPoolShim() {
    std::cout << "calling cuDeviceSetMemPoolShim" << std::endl;
}

void cuDeviceGetMemPoolShim() {
    std::cout << "calling cuDeviceGetMemPoolShim" << std::endl;
}

void cuFlushGPUDirectRDMAWritesShim() {
    std::cout << "calling cuFlushGPUDirectRDMAWritesShim" << std::endl;
}

void cuDevicePrimaryCtxRetainShim() {
    std::cout << "calling cuDevicePrimaryCtxRetainShim" << std::endl;
}

void cuDevicePrimaryCtxReleaseShim() {
    std::cout << "calling cuDevicePrimaryCtxReleaseShim" << std::endl;
}

void cuDevicePrimaryCtxSetFlagsShim() {
    std::cout << "calling cuDevicePrimaryCtxSetFlagsShim" << std::endl;
}

void cuDevicePrimaryCtxGetStateShim() {
    std::cout << "calling cuDevicePrimaryCtxGetStateShim" << std::endl;
}

void cuDevicePrimaryCtxResetShim() {
    std::cout << "calling cuDevicePrimaryCtxResetShim" << std::endl;
}

void cuCtxCreateShim() {
    std::cout << "calling cuCtxCreateShim" << std::endl;
}

void cuCtxGetFlagsShim() {
    std::cout << "calling cuCtxGetFlagsShim" << std::endl;
}

void cuCtxSetCurrentShim() {
    std::cout << "calling cuCtxSetCurrentShim" << std::endl;
}

void cuCtxGetCurrentShim() {
    std::cout << "calling cuCtxGetCurrentShim" << std::endl;
}

void cuCtxDetachShim() {
    std::cout << "calling cuCtxDetachShim" << std::endl;
}

void cuCtxGetApiVersionShim() {
    std::cout << "calling cuCtxGetApiVersionShim" << std::endl;
}

void cuCtxGetDeviceShim() {
    std::cout << "calling cuCtxGetDeviceShim" << std::endl;
}

void cuCtxGetLimitShim() {
    std::cout << "calling cuCtxGetLimitShim" << std::endl;
}

void cuCtxSetLimitShim() {
    std::cout << "calling cuCtxSetLimitShim" << std::endl;
}

void cuCtxGetCacheConfigShim() {
    std::cout << "calling cuCtxGetCacheConfigShim" << std::endl;
}

void cuCtxSetCacheConfigShim() {
    std::cout << "calling cuCtxSetCacheConfigShim" << std::endl;
}

void cuCtxGetSharedMemConfigShim() {
    std::cout << "calling cuCtxGetSharedMemConfigShim" << std::endl;
}

void cuCtxGetStreamPriorityRangeShim() {
    std::cout << "calling cuCtxGetStreamPriorityRangeShim" << std::endl;
}

void cuCtxSetSharedMemConfigShim() {
    std::cout << "calling cuCtxSetSharedMemConfigShim" << std::endl;
}

void cuCtxSynchronizeShim() {
    std::cout << "calling cuCtxSynchronizeShim" << std::endl;
}

void cuCtxResetPersistingL2CacheShim() {
    std::cout << "calling cuCtxResetPersistingL2CacheShim" << std::endl;
}

void cuCtxPopCurrentShim() {
    std::cout << "calling cuCtxPopCurrentShim" << std::endl;
}

void cuCtxPushCurrentShim() {
    std::cout << "calling cuCtxPushCurrentShim" << std::endl;
}

void cuModuleLoadShim() {
    std::cout << "calling cuModuleLoadShim" << std::endl;
}

void cuModuleLoadDataShim() {
    std::cout << "calling cuModuleLoadDataShim" << std::endl;
}

void cuModuleLoadFatBinaryShim() {
    std::cout << "calling cuModuleLoadFatBinaryShim" << std::endl;
}

void cuModuleUnloadShim() {
    std::cout << "calling cuModuleUnloadShim" << std::endl;
}

void cuModuleGetFunctionShim() {
    std::cout << "calling cuModuleGetFunctionShim" << std::endl;
}

void cuModuleGetGlobalShim() {
    std::cout << "calling cuModuleGetGlobalShim" << std::endl;
}

void cuModuleGetTexRefShim() {
    std::cout << "calling cuModuleGetTexRefShim" << std::endl;
}

void cuModuleGetSurfRefShim() {
    std::cout << "calling cuModuleGetSurfRefShim" << std::endl;
}

hipError_t cuModuleGetLoadingModeShim(CUmoduleLoadingMode *mode) {
    std::cout << "Client: calling cuModuleGetLoadingModeShim" << std::endl;

    if (sockfd < 0) {
        std::cerr << "Socket not connected." << std::endl;
        return hipErrorUnknown;
    }

    // Prepare the request ID for the server
    int request_id = rpc_start_request(RPC_cuModuleGetLoadingMode);
    if (request_id < 0) {
        std::cerr << "Failed to start request for cuModuleGetLoadingMode" << std::endl;
        return hipErrorUnknown;
    }

    // Wait for the server's response
    if (rpc_wait_for_response(request_id) < 0) {
        std::cerr << "Failed to wait for response from server" << std::endl;
        return hipErrorUnknown;
    }

    // Read the result code from the server
    hipError_t result;
    ssize_t bytes_read = rpc_read(&result, sizeof(hipError_t));
    if (bytes_read < 0) {
        std::cerr << "Failed to read result from server. Error: " << strerror(errno) << std::endl;
        return hipErrorUnknown;
    }

    // Check if the cuModuleGetLoadingMode call was successful
    if (result != hipSuccess) {
        std::cerr << "cuModuleGetLoadingMode call failed on the server. Error code: " << result << std::endl;
        return result;
    }

    // Read the loading mode from the server
    bytes_read = rpc_read(mode, sizeof(CUmoduleLoadingMode));
    if (bytes_read < 0) {
        std::cerr << "Failed to read loading mode from server. Error: " << strerror(errno) << std::endl;
        return hipErrorUnknown;
    }

    std::cout << "Client: Received loading mode from server: " << *mode << std::endl;

    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

void cuLibraryLoadDataShim() {
    std::cout << "calling cuLibraryLoadDataShim" << std::endl;
}

void cuLibraryLoadFromFileShim() {
    std::cout << "calling cuLibraryLoadFromFileShim" << std::endl;
}

void cuLibraryUnloadShim() {
    std::cout << "calling cuLibraryUnloadShim" << std::endl;
}

void cuLibraryGetKernelShim() {
    std::cout << "calling cuLibraryGetKernelShim" << std::endl;
}

void cuLibraryGetModuleShim() {
    std::cout << "calling cuLibraryGetModuleShim" << std::endl;
}

void cuKernelGetFunctionShim() {
    std::cout << "calling cuKernelGetFunctionShim" << std::endl;
}

void cuLibraryGetGlobalShim() {
    std::cout << "calling cuLibraryGetGlobalShim" << std::endl;
}

void cuLibraryGetManagedShim() {
    std::cout << "calling cuLibraryGetManagedShim" << std::endl;
}

void cuKernelGetAttributeShim() {
    std::cout << "calling cuKernelGetAttributeShim" << std::endl;
}

void cuKernelSetAttributeShim() {
    std::cout << "calling cuKernelSetAttributeShim" << std::endl;
}

void cuKernelSetCacheConfigShim() {
    std::cout << "calling cuKernelSetCacheConfigShim" << std::endl;
}

void cuLinkCreateShim() {
    std::cout << "calling cuLinkCreateShim" << std::endl;
}

void cuLinkAddDataShim() {
    std::cout << "calling cuLinkAddDataShim" << std::endl;
}

void cuLinkAddFileShim() {
    std::cout << "calling cuLinkAddFileShim" << std::endl;
}

void cuLinkCompleteShim() {
    std::cout << "calling cuLinkCompleteShim" << std::endl;
}

void cuLinkDestroyShim() {
    std::cout << "calling cuLinkDestroyShim" << std::endl;
}

void cuMemGetInfoShim() {
    std::cout << "calling cuMemGetInfoShim" << std::endl;
}

void cuMemAllocManagedShim() {
    std::cout << "calling cuMemAllocManagedShim" << std::endl;
}

void cuMemAllocShim() {
    std::cout << "calling cuMemAllocShim" << std::endl;
}

void cuMemAllocPitchShim() {
    std::cout << "calling cuMemAllocPitchShim" << std::endl;
}

void cuMemFreeShim() {
    std::cout << "calling cuMemFreeShim" << std::endl;
}

void cuMemGetAddressRangeShim() {
    std::cout << "calling cuMemGetAddressRangeShim" << std::endl;
}

void cuMemFreeHostShim() {
    std::cout << "calling cuMemFreeHostShim" << std::endl;
}

void cuMemHostAllocShim() {
    std::cout << "calling cuMemHostAllocShim" << std::endl;
}

void cuMemHostGetDevicePointerShim() {
    std::cout << "calling cuMemHostGetDevicePointerShim" << std::endl;
}

void cuMemHostGetFlagsShim() {
    std::cout << "calling cuMemHostGetFlagsShim" << std::endl;
}

void cuMemHostRegisterShim() {
    std::cout << "calling cuMemHostRegisterShim" << std::endl;
}

void cuMemHostUnregisterShim() {
    std::cout << "calling cuMemHostUnregisterShim" << std::endl;
}

void cuPointerGetAttributeShim() {
    std::cout << "calling cuPointerGetAttributeShim" << std::endl;
}

void cuPointerGetAttributesShim() {
    std::cout << "calling cuPointerGetAttributesShim" << std::endl;
}

void cuMemAllocAsyncShim() {
    std::cout << "calling cuMemAllocAsyncShim" << std::endl;
}

void cuMemAllocFromPoolAsyncShim() {
    std::cout << "calling cuMemAllocFromPoolAsyncShim" << std::endl;
}

void cuMemFreeAsyncShim() {
    std::cout << "calling cuMemFreeAsyncShim" << std::endl;
}

void cuMemPoolTrimToShim() {
    std::cout << "calling cuMemPoolTrimToShim" << std::endl;
}

void cuMemPoolSetAttributeShim() {
    std::cout << "calling cuMemPoolSetAttributeShim" << std::endl;
}

void cuMemPoolGetAttributeShim() {
    std::cout << "calling cuMemPoolGetAttributeShim" << std::endl;
}

void cuMemPoolSetAccessShim() {
    std::cout << "calling cuMemPoolSetAccessShim" << std::endl;
}

void cuMemPoolGetAccessShim() {
    std::cout << "calling cuMemPoolGetAccessShim" << std::endl;
}

void cuMemPoolCreateShim() {
    std::cout << "calling cuMemPoolCreateShim" << std::endl;
}

void cuMemPoolDestroyShim() {
    std::cout << "calling cuMemPoolDestroyShim" << std::endl;
}

void cuMemPoolExportToShareableHandleShim() {
    std::cout << "calling cuMemPoolExportToShareableHandleShim" << std::endl;
}

void cuMemPoolImportFromShareableHandleShim() {
    std::cout << "calling cuMemPoolImportFromShareableHandleShim" << std::endl;
}

void cuMemPoolExportPointerShim() {
    std::cout << "calling cuMemPoolExportPointerShim" << std::endl;
}

void cuMemPoolImportPointerShim() {
    std::cout << "calling cuMemPoolImportPointerShim" << std::endl;
}

void cuMemcpyShim() {
    std::cout << "calling cuMemcpyShim" << std::endl;
}

void cuMemcpyAsyncShim() {
    std::cout << "calling cuMemcpyAsyncShim" << std::endl;
}

void cuMemcpyPeerShim() {
    std::cout << "calling cuMemcpyPeerShim" << std::endl;
}

void cuMemcpyPeerAsyncShim() {
    std::cout << "calling cuMemcpyPeerAsyncShim" << std::endl;
}

void cuMemcpyHtoDShim() {
    std::cout << "calling cuMemcpyHtoDShim" << std::endl;
}

void cuMemcpyHtoDAsyncShim() {
    std::cout << "calling cuMemcpyHtoDAsyncShim" << std::endl;
}

void cuMemcpyDtoHShim() {
    std::cout << "calling cuMemcpyDtoHShim" << std::endl;
}

void cuMemcpyDtoHAsyncShim() {
    std::cout << "calling cuMemcpyDtoHAsyncShim" << std::endl;
}

void cuMemcpyDtoDShim() {
    std::cout << "calling cuMemcpyDtoDShim" << std::endl;
}

void cuMemcpyDtoDAsyncShim() {
    std::cout << "calling cuMemcpyDtoDAsyncShim" << std::endl;
}

void cuMemcpy2DUnalignedShim() {
    std::cout << "calling cuMemcpy2DUnalignedShim" << std::endl;
}

void cuMemcpy2DAsyncShim() {
    std::cout << "calling cuMemcpy2DAsyncShim" << std::endl;
}

void cuMemcpy3DShim() {
    std::cout << "calling cuMemcpy3DShim" << std::endl;
}

void cuMemcpy3DAsyncShim() {
    std::cout << "calling cuMemcpy3DAsyncShim" << std::endl;
}

void cuMemcpy3DPeerShim() {
    std::cout << "calling cuMemcpy3DPeerShim" << std::endl;
}

void cuMemcpy3DPeerAsyncShim() {
    std::cout << "calling cuMemcpy3DPeerAsyncShim" << std::endl;
}

void cuMemsetD8Shim() {
    std::cout << "calling cuMemsetD8Shim" << std::endl;
}

void cuMemsetD8AsyncShim() {
    std::cout << "calling cuMemsetD8AsyncShim" << std::endl;
}

void cuMemsetD2D8Shim() {
    std::cout << "calling cuMemsetD2D8Shim" << std::endl;
}

void cuMemsetD2D8AsyncShim() {
    std::cout << "calling cuMemsetD2D8AsyncShim" << std::endl;
}

void cuFuncSetCacheConfigShim() {
    std::cout << "calling cuFuncSetCacheConfigShim" << std::endl;
}

void cuFuncSetSharedMemConfigShim() {
    std::cout << "calling cuFuncSetSharedMemConfigShim" << std::endl;
}

void cuFuncGetAttributeShim() {
    std::cout << "calling cuFuncGetAttributeShim" << std::endl;
}

void cuFuncSetAttributeShim() {
    std::cout << "calling cuFuncSetAttributeShim" << std::endl;
}

void cuArrayCreateShim() {
    std::cout << "calling cuArrayCreateShim" << std::endl;
}

void cuArrayGetDescriptorShim() {
    std::cout << "calling cuArrayGetDescriptorShim" << std::endl;
}

void cuArrayGetSparsePropertiesShim() {
    std::cout << "calling cuArrayGetSparsePropertiesShim" << std::endl;
}

void cuArrayGetPlaneShim() {
    std::cout << "calling cuArrayGetPlaneShim" << std::endl;
}

void cuArray3DCreateShim() {
    std::cout << "calling cuArray3DCreateShim" << std::endl;
}

void cuArray3DGetDescriptorShim() {
    std::cout << "calling cuArray3DGetDescriptorShim" << std::endl;
}

void cuArrayDestroyShim() {
    std::cout << "calling cuArrayDestroyShim" << std::endl;
}

void cuMipmappedArrayCreateShim() {
    std::cout << "calling cuMipmappedArrayCreateShim" << std::endl;
}

void cuMipmappedArrayGetLevelShim() {
    std::cout << "calling cuMipmappedArrayGetLevelShim" << std::endl;
}

void cuMipmappedArrayGetSparsePropertiesShim() {
    std::cout << "calling cuMipmappedArrayGetSparsePropertiesShim" << std::endl;
}

void cuMipmappedArrayDestroyShim() {
    std::cout << "calling cuMipmappedArrayDestroyShim" << std::endl;
}

void cuArrayGetMemoryRequirementsShim() {
    std::cout << "calling cuArrayGetMemoryRequirementsShim" << std::endl;
}

void cuMipmappedArrayGetMemoryRequirementsShim() {
    std::cout << "calling cuMipmappedArrayGetMemoryRequirementsShim" << std::endl;
}

void cuTexObjectCreateShim() {
    std::cout << "calling cuTexObjectCreateShim" << std::endl;
}

void cuTexObjectDestroyShim() {
    std::cout << "calling cuTexObjectDestroyShim" << std::endl;
}

void cuTexObjectGetResourceDescShim() {
    std::cout << "calling cuTexObjectGetResourceDescShim" << std::endl;
}

void cuTexObjectGetTextureDescShim() {
    std::cout << "calling cuTexObjectGetTextureDescShim" << std::endl;
}

void cuTexObjectGetResourceViewDescShim() {
    std::cout << "calling cuTexObjectGetResourceViewDescShim" << std::endl;
}

void cuSurfObjectCreateShim() {
    std::cout << "calling cuSurfObjectCreateShim" << std::endl;
}

void cuSurfObjectDestroyShim() {
    std::cout << "calling cuSurfObjectDestroyShim" << std::endl;
}

void cuSurfObjectGetResourceDescShim() {
    std::cout << "calling cuSurfObjectGetResourceDescShim" << std::endl;
}

void cuImportExternalMemoryShim() {
    std::cout << "calling cuImportExternalMemoryShim" << std::endl;
}

void cuExternalMemoryGetMappedBufferShim() {
    std::cout << "calling cuExternalMemoryGetMappedBufferShim" << std::endl;
}

void cuExternalMemoryGetMappedMipmappedArrayShim() {
    std::cout << "calling cuExternalMemoryGetMappedMipmappedArrayShim" << std::endl;
}

void cuDestroyExternalMemoryShim() {
    std::cout << "calling cuDestroyExternalMemoryShim" << std::endl;
}

void cuImportExternalSemaphoreShim() {
    std::cout << "calling cuImportExternalSemaphoreShim" << std::endl;
}

void cuSignalExternalSemaphoresAsyncShim() {
    std::cout << "calling cuSignalExternalSemaphoresAsyncShim" << std::endl;
}

void cuWaitExternalSemaphoresAsyncShim() {
    std::cout << "calling cuWaitExternalSemaphoresAsyncShim" << std::endl;
}

void cuDestroyExternalSemaphoreShim() {
    std::cout << "calling cuDestroyExternalSemaphoreShim" << std::endl;
}

void cuDeviceGetNvSciSyncAttributesShim() {
    std::cout << "calling cuDeviceGetNvSciSyncAttributesShim" << std::endl;
}

void cuLaunchKernelShim() {
    std::cout << "calling cuLaunchKernelShim" << std::endl;
}

void cuLaunchCooperativeKernelShim() {
    std::cout << "calling cuLaunchCooperativeKernelShim" << std::endl;
}

void cuLaunchCooperativeKernelMultiDeviceShim() {
    std::cout << "calling cuLaunchCooperativeKernelMultiDeviceShim" << std::endl;
}

void cuLaunchHostFuncShim() {
    std::cout << "calling cuLaunchHostFuncShim" << std::endl;
}

void cuLaunchKernelExShim() {
    std::cout << "calling cuLaunchKernelExShim" << std::endl;
}

void cuEventCreateShim() {
    std::cout << "calling cuEventCreateShim" << std::endl;
}

void cuEventRecordShim() {
    std::cout << "calling cuEventRecordShim" << std::endl;
}

void cuEventRecordWithFlagsShim() {
    std::cout << "calling cuEventRecordWithFlagsShim" << std::endl;
}

void cuEventQueryShim() {
    std::cout << "calling cuEventQueryShim" << std::endl;
}

void cuEventSynchronizeShim() {
    std::cout << "calling cuEventSynchronizeShim" << std::endl;
}

void cuEventDestroyShim() {
    std::cout << "calling cuEventDestroyShim" << std::endl;
}

void cuEventElapsedTimeShim() {
    std::cout << "calling cuEventElapsedTimeShim" << std::endl;
}

void cuStreamWaitValue32Shim() {
    std::cout << "calling cuStreamWaitValue32Shim" << std::endl;
}

void cuStreamWriteValue32Shim() {
    std::cout << "calling cuStreamWriteValue32Shim" << std::endl;
}

void cuStreamWaitValue64Shim() {
    std::cout << "calling cuStreamWaitValue64Shim" << std::endl;
}

void cuStreamWriteValue64Shim() {
    std::cout << "calling cuStreamWriteValue64Shim" << std::endl;
}

void cuStreamBatchMemOpShim() {
    std::cout << "calling cuStreamBatchMemOpShim" << std::endl;
}

void cuStreamCreateShim() {
    std::cout << "calling cuStreamCreateShim" << std::endl;
}

void cuStreamCreateWithPriorityShim() {
    std::cout << "calling cuStreamCreateWithPriorityShim" << std::endl;
}

void cuStreamGetPriorityShim() {
    std::cout << "calling cuStreamGetPriorityShim" << std::endl;
}

void cuStreamGetFlagsShim() {
    std::cout << "calling cuStreamGetFlagsShim" << std::endl;
}

void cuStreamGetCtxShim() {
    std::cout << "calling cuStreamGetCtxShim" << std::endl;
}

void cuStreamGetIdShim() {
    std::cout << "calling cuStreamGetIdShim" << std::endl;
}

void cuStreamDestroyShim() {
    std::cout << "calling cuStreamDestroyShim" << std::endl;
}

void cuStreamWaitEventShim() {
    std::cout << "calling cuStreamWaitEventShim" << std::endl;
}

void cuStreamAddCallbackShim() {
    std::cout << "calling cuStreamAddCallbackShim" << std::endl;
}

void cuStreamSynchronizeShim() {
    std::cout << "calling cuStreamSynchronizeShim" << std::endl;
}

void cuStreamQueryShim() {
    std::cout << "calling cuStreamQueryShim" << std::endl;
}

void cuStreamAttachMemAsyncShim() {
    std::cout << "calling cuStreamAttachMemAsyncShim" << std::endl;
}

void cuStreamCopyAttributesShim() {
    std::cout << "calling cuStreamCopyAttributesShim" << std::endl;
}

void cuStreamGetAttributeShim() {
    std::cout << "calling cuStreamGetAttributeShim" << std::endl;
}

void cuStreamSetAttributeShim() {
    std::cout << "calling cuStreamSetAttributeShim" << std::endl;
}

void cuDeviceCanAccessPeerShim() {
    std::cout << "calling cuDeviceCanAccessPeerShim" << std::endl;
}

void cuCtxEnablePeerAccessShim() {
    std::cout << "calling cuCtxEnablePeerAccessShim" << std::endl;
}

void cuCtxDisablePeerAccessShim() {
    std::cout << "calling cuCtxDisablePeerAccessShim" << std::endl;
}

void cuIpcGetEventHandleShim() {
    std::cout << "calling cuIpcGetEventHandleShim" << std::endl;
}

void cuIpcOpenEventHandleShim() {
    std::cout << "calling cuIpcOpenEventHandleShim" << std::endl;
}

void cuIpcGetMemHandleShim() {
    std::cout << "calling cuIpcGetMemHandleShim" << std::endl;
}

void cuIpcOpenMemHandleShim() {
    std::cout << "calling cuIpcOpenMemHandleShim" << std::endl;
}

void cuIpcCloseMemHandleShim() {
    std::cout << "calling cuIpcCloseMemHandleShim" << std::endl;
}

void cuGLCtxCreateShim() {
    std::cout << "calling cuGLCtxCreateShim" << std::endl;
}

void cuGLInitShim() {
    std::cout << "calling cuGLInitShim" << std::endl;
}

void cuGLGetDevicesShim() {
    std::cout << "calling cuGLGetDevicesShim" << std::endl;
}

void cuGLRegisterBufferObjectShim() {
    std::cout << "calling cuGLRegisterBufferObjectShim" << std::endl;
}

void cuGLMapBufferObjectShim() {
    std::cout << "calling cuGLMapBufferObjectShim" << std::endl;
}

void cuGLMapBufferObjectAsyncShim() {
    std::cout << "calling cuGLMapBufferObjectAsyncShim" << std::endl;
}

void cuGLUnmapBufferObjectShim() {
    std::cout << "calling cuGLUnmapBufferObjectShim" << std::endl;
}

void cuGLUnmapBufferObjectAsyncShim() {
    std::cout << "calling cuGLUnmapBufferObjectAsyncShim" << std::endl;
}

void cuGLUnregisterBufferObjectShim() {
    std::cout << "calling cuGLUnregisterBufferObjectShim" << std::endl;
}

void cuGLSetBufferObjectMapFlagsShim() {
    std::cout << "calling cuGLSetBufferObjectMapFlagsShim" << std::endl;
}

void cuGraphicsGLRegisterImageShim() {
    std::cout << "calling cuGraphicsGLRegisterImageShim" << std::endl;
}

void cuGraphicsGLRegisterBufferShim() {
    std::cout << "calling cuGraphicsGLRegisterBufferShim" << std::endl;
}

void cuGraphicsEGLRegisterImageShim() {
    std::cout << "calling cuGraphicsEGLRegisterImageShim" << std::endl;
}

void cuEGLStreamConsumerConnectShim() {
    std::cout << "calling cuEGLStreamConsumerConnectShim" << std::endl;
}

void cuEGLStreamConsumerDisconnectShim() {
    std::cout << "calling cuEGLStreamConsumerDisconnectShim" << std::endl;
}

void cuEGLStreamConsumerAcquireFrameShim() {
    std::cout << "calling cuEGLStreamConsumerAcquireFrameShim" << std::endl;
}

void cuEGLStreamConsumerReleaseFrameShim() {
    std::cout << "calling cuEGLStreamConsumerReleaseFrameShim" << std::endl;
}

void cuEGLStreamProducerConnectShim() {
    std::cout << "calling cuEGLStreamProducerConnectShim" << std::endl;
}

void cuEGLStreamProducerDisconnectShim() {
    std::cout << "calling cuEGLStreamProducerDisconnectShim" << std::endl;
}

void cuEGLStreamProducerPresentFrameShim() {
    std::cout << "calling cuEGLStreamProducerPresentFrameShim" << std::endl;
}

void cuEGLStreamProducerReturnFrameShim() {
    std::cout << "calling cuEGLStreamProducerReturnFrameShim" << std::endl;
}

void cuGraphicsResourceGetMappedEglFrameShim() {
    std::cout << "calling cuGraphicsResourceGetMappedEglFrameShim" << std::endl;
}

void cuGraphicsUnregisterResourceShim() {
    std::cout << "calling cuGraphicsUnregisterResourceShim" << std::endl;
}

void cuGraphicsMapResourcesShim() {
    std::cout << "calling cuGraphicsMapResourcesShim" << std::endl;
}

void cuGraphicsUnmapResourcesShim() {
    std::cout << "calling cuGraphicsUnmapResourcesShim" << std::endl;
}

void cuGraphicsResourceSetMapFlagsShim() {
    std::cout << "calling cuGraphicsResourceSetMapFlagsShim" << std::endl;
}

void cuGraphicsSubResourceGetMappedArrayShim() {
    std::cout << "calling cuGraphicsSubResourceGetMappedArrayShim" << std::endl;
}

void cuGraphicsResourceGetMappedMipmappedArrayShim() {
    std::cout << "calling cuGraphicsResourceGetMappedMipmappedArrayShim" << std::endl;
}

void cuProfilerInitializeShim() {
    std::cout << "calling cuProfilerInitializeShim" << std::endl;
}

void cuProfilerStartShim() {
    std::cout << "calling cuProfilerStartShim" << std::endl;
}

void cuProfilerStopShim() {
    std::cout << "calling cuProfilerStopShim" << std::endl;
}

void cuVDPAUGetDeviceShim() {
    std::cout << "calling cuVDPAUGetDeviceShim" << std::endl;
}

void cuVDPAUCtxCreateShim() {
    std::cout << "calling cuVDPAUCtxCreateShim" << std::endl;
}

void cuGraphicsVDPAURegisterVideoSurfaceShim() {
    std::cout << "calling cuGraphicsVDPAURegisterVideoSurfaceShim" << std::endl;
}

void cuGraphicsVDPAURegisterOutputSurfaceShim() {
    std::cout << "calling cuGraphicsVDPAURegisterOutputSurfaceShim" << std::endl;
}

hipError_t cuGetExportTableShim(void **ppExportTable, const hipUUID *pTableUuid) {
    std::cout << "calling cuGetExportTableShim" << std::endl;

    if (sockfd < 0) {
        std::cerr << "Socket not connected." << std::endl;
        return hipErrorUnknown;
    }

    // Start the request to the server
    int request_id = rpc_start_request(RPC_cuGetExportTable);
    if (request_id < 0) {
        std::cerr << "Failed to start request for cuGetExportTable" << std::endl;
        return hipErrorUnknown;
    }

    // Check if pTableUuid is valid
    if (pTableUuid == nullptr) {
        std::cerr << "Invalid UUID pointer provided to cuGetExportTableShim" << std::endl;
        return hipErrorInvalidValue;
    }

    // Send the UUID to the server
    if (rpc_write(pTableUuid, sizeof(hipUUID)) < 0) {
        std::cerr << "Failed to write UUID to server" << std::endl;
        return hipErrorUnknown;
    }

    // Wait for the server response
    if (rpc_wait_for_response(request_id) < 0) {
        std::cerr << "Failed to wait for response from server" << std::endl;
        return hipErrorUnknown;
    }

    // Read the result code from the server
    hipError_t result;
    if (rpc_read(&result, sizeof(hipError_t)) < 0) {
        std::cerr << "Failed to read result from server" << std::endl;
        return hipErrorUnknown;
    }

    // Check if the cuGetExportTable call was successful
    if (result != hipSuccess) {
        std::cerr << "cuGetExportTable call failed on the server. Error code: " << result << std::endl;
        return result;
    }

    // Read the export table pointer from the server
    if (rpc_read(ppExportTable, sizeof(void *)) < 0) {
        std::cerr << "Failed to read export table pointer from server" << std::endl;
        return hipErrorUnknown;
    }

    std::cout << "Client: Received export table pointer from server: " << *ppExportTable << std::endl;

    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

void cuOccupancyMaxActiveBlocksPerMultiprocessorWithFlagsShim() {
    std::cout << "calling cuOccupancyMaxActiveBlocksPerMultiprocessorWithFlagsShim" << std::endl;
}

void cuOccupancyAvailableDynamicSMemPerBlockShim() {
    std::cout << "calling cuOccupancyAvailableDynamicSMemPerBlockShim" << std::endl;
}

void cuOccupancyMaxPotentialClusterSizeShim() {
    std::cout << "calling cuOccupancyMaxPotentialClusterSizeShim" << std::endl;
}

void cuOccupancyMaxActiveClustersShim() {
    std::cout << "calling cuOccupancyMaxActiveClustersShim" << std::endl;
}

void cuMemAdviseShim() {
    std::cout << "calling cuMemAdviseShim" << std::endl;
}

void cuMemPrefetchAsyncShim() {
    std::cout << "calling cuMemPrefetchAsyncShim" << std::endl;
}

void cuMemRangeGetAttributeShim() {
    std::cout << "calling cuMemRangeGetAttributeShim" << std::endl;
}

void cuMemRangeGetAttributesShim() {
    std::cout << "calling cuMemRangeGetAttributesShim" << std::endl;
}

void cuGetErrorNameShim() {
    std::cout << "calling cuGetErrorNameShim" << std::endl;
}

void cuGraphCreateShim() {
    std::cout << "calling cuGraphCreateShim" << std::endl;
}

void cuGraphAddKernelNodeShim() {
    std::cout << "calling cuGraphAddKernelNodeShim" << std::endl;
}

void cuGraphKernelNodeGetParamsShim() {
    std::cout << "calling cuGraphKernelNodeGetParamsShim" << std::endl;
}

void cuGraphKernelNodeSetParamsShim() {
    std::cout << "calling cuGraphKernelNodeSetParamsShim" << std::endl;
}

void cuGraphAddMemcpyNodeShim() {
    std::cout << "calling cuGraphAddMemcpyNodeShim" << std::endl;
}

void cuGraphMemcpyNodeGetParamsShim() {
    std::cout << "calling cuGraphMemcpyNodeGetParamsShim" << std::endl;
}

void cuGraphMemcpyNodeSetParamsShim() {
    std::cout << "calling cuGraphMemcpyNodeSetParamsShim" << std::endl;
}

void cuGraphAddMemsetNodeShim() {
    std::cout << "calling cuGraphAddMemsetNodeShim" << std::endl;
}

void cuGraphMemsetNodeGetParamsShim() {
    std::cout << "calling cuGraphMemsetNodeGetParamsShim" << std::endl;
}

void cuGraphMemsetNodeSetParamsShim() {
    std::cout << "calling cuGraphMemsetNodeSetParamsShim" << std::endl;
}

void cuGraphAddHostNodeShim() {
    std::cout << "calling cuGraphAddHostNodeShim" << std::endl;
}

void cuGraphHostNodeGetParamsShim() {
    std::cout << "calling cuGraphHostNodeGetParamsShim" << std::endl;
}

void cuGraphHostNodeSetParamsShim() {
    std::cout << "calling cuGraphHostNodeSetParamsShim" << std::endl;
}

void cuGraphAddChildGraphNodeShim() {
    std::cout << "calling cuGraphAddChildGraphNodeShim" << std::endl;
}

void cuGraphChildGraphNodeGetGraphShim() {
    std::cout << "calling cuGraphChildGraphNodeGetGraphShim" << std::endl;
}

void cuGraphAddEmptyNodeShim() {
    std::cout << "calling cuGraphAddEmptyNodeShim" << std::endl;
}

void cuGraphAddEventRecordNodeShim() {
    std::cout << "calling cuGraphAddEventRecordNodeShim" << std::endl;
}

void cuGraphEventRecordNodeGetEventShim() {
    std::cout << "calling cuGraphEventRecordNodeGetEventShim" << std::endl;
}

void cuGraphEventRecordNodeSetEventShim() {
    std::cout << "calling cuGraphEventRecordNodeSetEventShim" << std::endl;
}

void cuGraphAddEventWaitNodeShim() {
    std::cout << "calling cuGraphAddEventWaitNodeShim" << std::endl;
}

void cuGraphEventWaitNodeGetEventShim() {
    std::cout << "calling cuGraphEventWaitNodeGetEventShim" << std::endl;
}

void cuGraphEventWaitNodeSetEventShim() {
    std::cout << "calling cuGraphEventWaitNodeSetEventShim" << std::endl;
}

void cuGraphAddExternalSemaphoresSignalNodeShim() {
    std::cout << "calling cuGraphAddExternalSemaphoresSignalNodeShim" << std::endl;
}

void cuGraphExternalSemaphoresSignalNodeGetParamsShim() {
    std::cout << "calling cuGraphExternalSemaphoresSignalNodeGetParamsShim" << std::endl;
}

void cuGraphExternalSemaphoresSignalNodeSetParamsShim() {
    std::cout << "calling cuGraphExternalSemaphoresSignalNodeSetParamsShim" << std::endl;
}

void cuGraphAddExternalSemaphoresWaitNodeShim() {
    std::cout << "calling cuGraphAddExternalSemaphoresWaitNodeShim" << std::endl;
}

void cuGraphExternalSemaphoresWaitNodeGetParamsShim() {
    std::cout << "calling cuGraphExternalSemaphoresWaitNodeGetParamsShim" << std::endl;
}

void cuGraphExternalSemaphoresWaitNodeSetParamsShim() {
    std::cout << "calling cuGraphExternalSemaphoresWaitNodeSetParamsShim" << std::endl;
}

void cuGraphExecExternalSemaphoresSignalNodeSetParamsShim() {
    std::cout << "calling cuGraphExecExternalSemaphoresSignalNodeSetParamsShim" << std::endl;
}

void cuGraphExecExternalSemaphoresWaitNodeSetParamsShim() {
    std::cout << "calling cuGraphExecExternalSemaphoresWaitNodeSetParamsShim" << std::endl;
}

void cuGraphAddMemAllocNodeShim() {
    std::cout << "calling cuGraphAddMemAllocNodeShim" << std::endl;
}

void cuGraphMemAllocNodeGetParamsShim() {
    std::cout << "calling cuGraphMemAllocNodeGetParamsShim" << std::endl;
}

void cuGraphAddMemFreeNodeShim() {
    std::cout << "calling cuGraphAddMemFreeNodeShim" << std::endl;
}

void cuGraphMemFreeNodeGetParamsShim() {
    std::cout << "calling cuGraphMemFreeNodeGetParamsShim" << std::endl;
}

void cuDeviceGraphMemTrimShim() {
    std::cout << "calling cuDeviceGraphMemTrimShim" << std::endl;
}

void cuDeviceGetGraphMemAttributeShim() {
    std::cout << "calling cuDeviceGetGraphMemAttributeShim" << std::endl;
}

void cuDeviceSetGraphMemAttributeShim() {
    std::cout << "calling cuDeviceSetGraphMemAttributeShim" << std::endl;
}

void cuGraphCloneShim() {
    std::cout << "calling cuGraphCloneShim" << std::endl;
}

void cuGraphNodeFindInCloneShim() {
    std::cout << "calling cuGraphNodeFindInCloneShim" << std::endl;
}

void cuGraphNodeGetTypeShim() {
    std::cout << "calling cuGraphNodeGetTypeShim" << std::endl;
}

void cuGraphGetNodesShim() {
    std::cout << "calling cuGraphGetNodesShim" << std::endl;
}

void cuGraphGetRootNodesShim() {
    std::cout << "calling cuGraphGetRootNodesShim" << std::endl;
}

void cuGraphGetEdgesShim() {
    std::cout << "calling cuGraphGetEdgesShim" << std::endl;
}

void cuGraphNodeGetDependenciesShim() {
    std::cout << "calling cuGraphNodeGetDependenciesShim" << std::endl;
}

void cuGraphNodeGetDependentNodesShim() {
    std::cout << "calling cuGraphNodeGetDependentNodesShim" << std::endl;
}

void cuGraphAddDependenciesShim() {
    std::cout << "calling cuGraphAddDependenciesShim" << std::endl;
}

void cuGraphRemoveDependenciesShim() {
    std::cout << "calling cuGraphRemoveDependenciesShim" << std::endl;
}

void cuGraphDestroyNodeShim() {
    std::cout << "calling cuGraphDestroyNodeShim" << std::endl;
}

void cuGraphInstantiateShim() {
    std::cout << "calling cuGraphInstantiateShim" << std::endl;
}

void cuGraphUploadShim() {
    std::cout << "calling cuGraphUploadShim" << std::endl;
}

void cuGraphLaunchShim() {
    std::cout << "calling cuGraphLaunchShim" << std::endl;
}

void cuGraphExecDestroyShim() {
    std::cout << "calling cuGraphExecDestroyShim" << std::endl;
}

void cuGraphDestroyShim() {
    std::cout << "calling cuGraphDestroyShim" << std::endl;
}

void cuStreamBeginCaptureShim() {
    std::cout << "calling cuStreamBeginCaptureShim" << std::endl;
}

void cuStreamEndCaptureShim() {
    std::cout << "calling cuStreamEndCaptureShim" << std::endl;
}

void cuStreamIsCapturingShim() {
    std::cout << "calling cuStreamIsCapturingShim" << std::endl;
}

void cuStreamGetCaptureInfoShim() {
    std::cout << "calling cuStreamGetCaptureInfoShim" << std::endl;
}

void cuStreamUpdateCaptureDependenciesShim() {
    std::cout << "calling cuStreamUpdateCaptureDependenciesShim" << std::endl;
}

void cuGraphExecKernelNodeSetParamsShim() {
    std::cout << "calling cuGraphExecKernelNodeSetParamsShim" << std::endl;
}

void cuGraphExecMemcpyNodeSetParamsShim() {
    std::cout << "calling cuGraphExecMemcpyNodeSetParamsShim" << std::endl;
}

void cuGraphExecMemsetNodeSetParamsShim() {
    std::cout << "calling cuGraphExecMemsetNodeSetParamsShim" << std::endl;
}

void cuGraphExecHostNodeSetParamsShim() {
    std::cout << "calling cuGraphExecHostNodeSetParamsShim" << std::endl;
}

void cuGraphExecChildGraphNodeSetParamsShim() {
    std::cout << "calling cuGraphExecChildGraphNodeSetParamsShim" << std::endl;
}

void cuGraphExecEventRecordNodeSetEventShim() {
    std::cout << "calling cuGraphExecEventRecordNodeSetEventShim" << std::endl;
}

void cuGraphExecEventWaitNodeSetEventShim() {
    std::cout << "calling cuGraphExecEventWaitNodeSetEventShim" << std::endl;
}

void cuThreadExchangeStreamCaptureModeShim() {
    std::cout << "calling cuThreadExchangeStreamCaptureModeShim" << std::endl;
}

void cuGraphExecUpdateShim() {
    std::cout << "calling cuGraphExecUpdateShim" << std::endl;
}

void cuGraphKernelNodeCopyAttributesShim() {
    std::cout << "calling cuGraphKernelNodeCopyAttributesShim" << std::endl;
}

void cuGraphKernelNodeGetAttributeShim() {
    std::cout << "calling cuGraphKernelNodeGetAttributeShim" << std::endl;
}

void cuGraphKernelNodeSetAttributeShim() {
    std::cout << "calling cuGraphKernelNodeSetAttributeShim" << std::endl;
}

void cuGraphDebugDotPrintShim() {
    std::cout << "calling cuGraphDebugDotPrintShim" << std::endl;
}

void cuUserObjectCreateShim() {
    std::cout << "calling cuUserObjectCreateShim" << std::endl;
}

void cuUserObjectRetainShim() {
    std::cout << "calling cuUserObjectRetainShim" << std::endl;
}

void cuUserObjectReleaseShim() {
    std::cout << "calling cuUserObjectReleaseShim" << std::endl;
}

void cuGraphRetainUserObjectShim() {
    std::cout << "calling cuGraphRetainUserObjectShim" << std::endl;
}

void cuGraphReleaseUserObjectShim() {
    std::cout << "calling cuGraphReleaseUserObjectShim" << std::endl;
}

void cuGraphNodeSetEnabledShim() {
    std::cout << "calling cuGraphNodeSetEnabledShim" << std::endl;
}

void cuGraphNodeGetEnabledShim() {
    std::cout << "calling cuGraphNodeGetEnabledShim" << std::endl;
}

void cuGraphInstantiateWithParamsShim() {
    std::cout << "calling cuGraphInstantiateWithParamsShim" << std::endl;
}

void cuGraphExecGetFlagsShim() {
    std::cout << "calling cuGraphExecGetFlagsShim" << std::endl;
}

// hipError_t hipGetDeviceCount(int* count) {
//     std::cout << "calling hipGetDeviceCount" << std::endl;
    
//     // Return a dummy value for testing
//     *count = 0;  // Set to 0 or any desired value for testing

//     // Return success
//     return hipSuccess;
// }

hipError_t hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(int *numBlocks, hipFunction_t func, int blockSize, size_t dynamicSMemSize)
{
    
    int request_id = rpc_start_request(RPC_cuOccupancyMaxActiveBlocksPerMultiprocessor);
    if (request_id < 0 ||
        rpc_write(&func, sizeof(hipFunction_t)) < 0 ||
        rpc_write(&blockSize, sizeof(int)) < 0 ||
        rpc_write(&dynamicSMemSize, sizeof(size_t)) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(numBlocks, sizeof(int)) < 0)
        return hipErrorUnknown;
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

hipError_t hipDrvLaunchKernelEx(const HIP_LAUNCH_CONFIG *config, hipFunction_t f, void **kernelParams, void **extra)
{
    
    // Start the RPC request for hipDrvLaunchKernelEx
    int request_id = rpc_start_request(RPC_cuLaunchKernelEx);

    // Error handling for request initiation
    if (request_id < 0)
        return hipErrorUnknown;

    // Write config to the request
    if (rpc_write(config, sizeof(HIP_LAUNCH_CONFIG)) < 0)
        return hipErrorUnknown;

    // Write hipFunction_t f to the request
    if (rpc_write(&f, sizeof(hipFunction_t)) < 0)
        return hipErrorUnknown;

    // Write kernel parameters to the request
    size_t kernelParamsSize = sizeof(void *) * config->gridDimX; // Adjust to the appropriate size as per your logic
    if (rpc_write(kernelParams, kernelParamsSize) < 0)
        return hipErrorUnknown;

    // Write extra parameters to the request
    size_t extraSize = sizeof(void *) * config->gridDimX; // Adjust to the appropriate size as per your logic
    if (rpc_write(extra, extraSize) < 0)
        return hipErrorUnknown;

    // Wait for the response
    if (rpc_wait_for_response(request_id) < 0)
        return hipErrorUnknown;

    // Return the result from the response
    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

// hipError_t hipMemcpyDtoH(void *dstHost, hipDeviceptr_t srcDevice, size_t ByteCount)
// {
//     int request_id = rpc_start_request(RPC_cuMemcpyDtoH_v2);

//     if (request_id < 0 ||
//         rpc_write(dstHost, sizeof(void*)) < 0 ||
//         rpc_write(&srcDevice, sizeof(hipDeviceptr_t)) < 0 ||
//         rpc_write(&ByteCount, sizeof(size_t)) < 0 ||
//         rpc_wait_for_response(request_id) < 0 ||
//         rpc_read(dstHost, ByteCount) < 0)
//     {
//         return hipErrorUnknown;
//     }

//     return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
// }

hipError_t hipModuleGetGlobal(hipDeviceptr_t *dptr, size_t *bytes, hipModule_t hmod, const char *name)
{
    int request_id = rpc_start_request(RPC_cuModuleGetGlobal_v2);

    if (request_id < 0 ||
        rpc_write(&hmod, sizeof(hipModule_t)) < 0 ||
        rpc_write(name, strlen(name) + 1) < 0 ||
        rpc_wait_for_response(request_id) < 0 ||
        rpc_read(dptr, sizeof(hipDeviceptr_t)) < 0 ||
        rpc_read(bytes, sizeof(size_t)) < 0)
    {
        return hipErrorUnknown;
    }

    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

// Map of symbols to their corresponding function pointers
std::unordered_map<std::string, void (*)()> cucudaFunctionMap;

hipError_t cuDriverGetVersion_handler(int *driverVersion) {
    if (driverVersion == nullptr) {
        return hipErrorInvalidValue;
    }

    // Start the RPC request for hipDriverGetVersion
    int request_id = rpc_start_request(RPC_cuDriverGetVersion);
    if (request_id < 0) {
        return hipErrorUnknown;
    }

    // Wait for the server response
    if (rpc_wait_for_response(request_id) < 0) {
        return hipErrorUnknown;
    }

    // Read the result code from the server
    hipError_t result;
    if (rpc_read(&result, sizeof(hipError_t)) < 0) {
        return hipErrorUnknown;
    }

    // If the result indicates an error, return it directly
    if (result != hipSuccess) {
        return result;
    }

    // Read the driver version from the server
    if (rpc_read(driverVersion, sizeof(int)) < 0) {
        return hipErrorUnknown;
    }

    std::cout << "Client: Received driver version from server: " << *driverVersion << std::endl;

    return rpc_get_return<hipError_t>(request_id, hipErrorUnknown);
}

void cuGraphInstantiateWithParams_ptszShim() {
    std::cout << "Client: calling cuGraphInstantiateWithParams_ptsz" << std::endl;
}

void cuGraphInstantiateWithFlagsShim() {
    std::cout << "Client: calling hipGraphInstantiateWithFlags" << std::endl;
}

void cuEGLStreamConsumerConnectWithFlagsShim() {
    std::cout << "Client: calling cuEGLStreamConsumerConnectWithFlags" << std::endl;
}

extern "C" void cuGraphicsResourceGetMappedPointerShim() {
    std::cout << "Client: calling cuGraphicsResourceGetMappedPointerShim" << std::endl;
}

extern "C" void cuGetErrorStringShim() {
    std::cout << "Client: calling cuGetErrorStringShim" << std::endl;
}

extern "C" void cuLinkAddData_v2Shim() {
    std::cout << "Client: calling cuLinkAddData_v2Shim" << std::endl;
}

extern "C" void cuModuleLoadDataExShim() {
    std::cout << "Client: calling cuModuleLoadDataExShim" << std::endl;
}

extern "C" void cuLinkAddFile_v2Shim() {
    std::cout << "Client: calling cuLinkAddFile_v2Shim" << std::endl;
}

extern "C" void cuMemcpyDtoH_v2Shim() {
    std::cout << "Client: calling cuMemcpyDtoH_v2Shim" << std::endl;
}

extern "C" void cuOccupancyMaxActiveBlocksPerMultiprocessorShim() {
    std::cout << "Client: calling cuOccupancyMaxActiveBlocksPerMultiprocessorShim" << std::endl;
}

extern "C" void cuModuleGetGlobal_v2Shim() {
    std::cout << "Client: calling cuModuleGetGlobal_v2Shim" << std::endl;
}

void noOpFunction() {
    // Do nothing
}

std::unordered_map<std::string, void *> cudaFunctionMap;

void initCudaFunctionMap() {
    cudaFunctionMap["hipInit"] = (void *)hipInit;
    cudaFunctionMap["hipGetProcAddress"] = (void *)hipGetProcAddress;
    cudaFunctionMap["hipDriverGetVersion"] = (void *)cuDriverGetVersion_handler;
    cudaFunctionMap["hipGetDeviceCount"] = (void *)hipGetDeviceCount;
    cudaFunctionMap["hipDeviceGet"] = (void *)cuDeviceGetShim;
    cudaFunctionMap["hipGetDeviceCount"] = (void *)cuDeviceGetCountShim;
    cudaFunctionMap["hipDeviceGetName"] = (void *)cuDeviceGetNameShim;
    cudaFunctionMap["hipDeviceTotalMem"] = (void *)cuDeviceTotalMemShim;
    cudaFunctionMap["hipDeviceGetAttribute"] = (void *)cuDeviceGetAttributeShim;
    cudaFunctionMap["hipDeviceGetP2PAttribute"] = (void *)cuDeviceGetP2PAttributeShim;
    cudaFunctionMap["hipDeviceGetByPCIBusId"] = (void *)cuDeviceGetByPCIBusIdShim;
    cudaFunctionMap["hipDeviceGetPCIBusId"] = (void *)cuDeviceGetPCIBusIdShim;
    cudaFunctionMap["hipDeviceGetUuid"] = (void *)cuDeviceGetUuidShim;
    cudaFunctionMap["cuDeviceGetTexture1DLinearMaxWidth"] = (void *)cuDeviceGetTexture1DLinearMaxWidthShim;
    cudaFunctionMap["hipDeviceGetDefaultMemPool"] = (void *)cuDeviceGetDefaultMemPoolShim;
    cudaFunctionMap["hipDeviceSetMemPool"] = (void *)cuDeviceSetMemPoolShim;
    cudaFunctionMap["hipDeviceGetMemPool"] = (void *)cuDeviceGetMemPoolShim;
    cudaFunctionMap["cuFlushGPUDirectRDMAWrites"] = (void *)cuFlushGPUDirectRDMAWritesShim;
    cudaFunctionMap["hipDevicePrimaryCtxRetain"] = (void *)cuDevicePrimaryCtxRetainShim;
    cudaFunctionMap["hipDevicePrimaryCtxRelease"] = (void *)cuDevicePrimaryCtxReleaseShim;
    cudaFunctionMap["hipDevicePrimaryCtxSetFlags"] = (void *)cuDevicePrimaryCtxSetFlagsShim;
    cudaFunctionMap["hipDevicePrimaryCtxGetState"] = (void *)cuDevicePrimaryCtxGetStateShim;
    cudaFunctionMap["hipDevicePrimaryCtxReset"] = (void *)cuDevicePrimaryCtxResetShim;
    cudaFunctionMap["hipCtxCreate"] = (void *)cuCtxCreateShim;
    cudaFunctionMap["hipCtxGetFlags"] = (void *)cuCtxGetFlagsShim;
    cudaFunctionMap["hipCtxSetCurrent"] = (void *)cuCtxSetCurrentShim;
    cudaFunctionMap["hipCtxGetCurrent"] = (void *)cuCtxGetCurrentShim;
    cudaFunctionMap["cuCtxDetach"] = (void *)cuCtxDetachShim;
    cudaFunctionMap["hipCtxGetApiVersion"] = (void *)cuCtxGetApiVersionShim;
    cudaFunctionMap["hipCtxGetDevice"] = (void *)cuCtxGetDeviceShim;
    cudaFunctionMap["hipDeviceGetLimit"] = (void *)cuCtxGetLimitShim;
    cudaFunctionMap["hipDeviceSetLimit"] = (void *)cuCtxSetLimitShim;
    cudaFunctionMap["hipCtxGetCacheConfig"] = (void *)cuCtxGetCacheConfigShim;
    cudaFunctionMap["hipCtxSetCacheConfig"] = (void *)cuCtxSetCacheConfigShim;
    cudaFunctionMap["hipCtxGetSharedMemConfig"] = (void *)cuCtxGetSharedMemConfigShim;
    cudaFunctionMap["hipDeviceGetStreamPriorityRange"] = (void *)cuCtxGetStreamPriorityRangeShim;
    cudaFunctionMap["hipCtxSetSharedMemConfig"] = (void *)cuCtxSetSharedMemConfigShim;
    cudaFunctionMap["hipCtxSynchronize"] = (void *)cuCtxSynchronizeShim;
    cudaFunctionMap["cuCtxResetPersistingL2Cache"] = (void *)cuCtxResetPersistingL2CacheShim;
    cudaFunctionMap["hipCtxPopCurrent"] = (void *)cuCtxPopCurrentShim;
    cudaFunctionMap["hipCtxPushCurrent"] = (void *)cuCtxPushCurrentShim;
    cudaFunctionMap["hipModuleLoad"] = (void *)cuModuleLoadShim;
    cudaFunctionMap["hipModuleLoadData"] = (void *)cuModuleLoadDataShim;
    cudaFunctionMap["hipModuleLoadFatBinary"] = (void *)cuModuleLoadFatBinaryShim;
    cudaFunctionMap["hipModuleUnload"] = (void *)cuModuleUnloadShim;
    cudaFunctionMap["hipModuleGetFunction"] = (void *)cuModuleGetFunctionShim;
    cudaFunctionMap["hipModuleGetGlobal"] = (void *)cuModuleGetGlobalShim;
    cudaFunctionMap["hipModuleGetTexRef"] = (void *)cuModuleGetTexRefShim;
    cudaFunctionMap["cuModuleGetSurfRef"] = (void *)cuModuleGetSurfRefShim;
    cudaFunctionMap["cuModuleGetLoadingMode"] = (void *)cuModuleGetLoadingModeShim;
    cudaFunctionMap["hipLibraryLoadData"] = (void *)cuLibraryLoadDataShim;
    cudaFunctionMap["hipLibraryLoadFromFile"] = (void *)cuLibraryLoadFromFileShim;
    cudaFunctionMap["hipLibraryUnload"] = (void *)cuLibraryUnloadShim;
    cudaFunctionMap["hipLibraryGetKernel"] = (void *)cuLibraryGetKernelShim;
    cudaFunctionMap["cuLibraryGetModule"] = (void *)cuLibraryGetModuleShim;
    cudaFunctionMap["cuKernelGetFunction"] = (void *)cuKernelGetFunctionShim;
    cudaFunctionMap["cuLibraryGetGlobal"] = (void *)cuLibraryGetGlobalShim;
    cudaFunctionMap["cuLibraryGetManaged"] = (void *)cuLibraryGetManagedShim;
    cudaFunctionMap["cuKernelGetAttribute"] = (void *)cuKernelGetAttributeShim;
    cudaFunctionMap["cuKernelSetAttribute"] = (void *)cuKernelSetAttributeShim;
    cudaFunctionMap["cuKernelSetCacheConfig"] = (void *)cuKernelSetCacheConfigShim;
    cudaFunctionMap["hiprtcLinkCreate"] = (void *)cuLinkCreateShim;
    cudaFunctionMap["hiprtcLinkAddData"] = (void *)cuLinkAddDataShim;
    cudaFunctionMap["hiprtcLinkAddFile"] = (void *)cuLinkAddFileShim;
    cudaFunctionMap["hiprtcLinkComplete"] = (void *)cuLinkCompleteShim;
    cudaFunctionMap["hiprtcLinkDestroy"] = (void *)cuLinkDestroyShim;
    cudaFunctionMap["hipMemGetInfo"] = (void *)cuMemGetInfoShim;
    cudaFunctionMap["hipMallocManaged"] = (void *)cuMemAllocManagedShim;
    cudaFunctionMap["hipMalloc"] = (void *)cuMemAllocShim;
    cudaFunctionMap["hipMemAllocPitch"] = (void *)cuMemAllocPitchShim;
    cudaFunctionMap["hipFree"] = (void *)cuMemFreeShim;
    cudaFunctionMap["hipMemGetAddressRange"] = (void *)cuMemGetAddressRangeShim;
    cudaFunctionMap["hipHostFree"] = (void *)cuMemFreeHostShim;
    cudaFunctionMap["hipHostAlloc"] = (void *)cuMemHostAllocShim;
    cudaFunctionMap["hipHostGetDevicePointer"] = (void *)cuMemHostGetDevicePointerShim;
    cudaFunctionMap["hipHostGetFlags"] = (void *)cuMemHostGetFlagsShim;
    cudaFunctionMap["hipHostRegister"] = (void *)cuMemHostRegisterShim;
    cudaFunctionMap["hipHostUnregister"] = (void *)cuMemHostUnregisterShim;
    cudaFunctionMap["hipPointerGetAttribute"] = (void *)cuPointerGetAttributeShim;
    cudaFunctionMap["hipDrvPointerGetAttributes"] = (void *)cuPointerGetAttributesShim;
    cudaFunctionMap["hipMallocAsync"] = (void *)cuMemAllocAsyncShim;
    cudaFunctionMap["hipMallocFromPoolAsync"] = (void *)cuMemAllocFromPoolAsyncShim;
    cudaFunctionMap["hipFreeAsync"] = (void *)cuMemFreeAsyncShim;
    cudaFunctionMap["hipMemPoolTrimTo"] = (void *)cuMemPoolTrimToShim;
    cudaFunctionMap["hipMemPoolSetAttribute"] = (void *)cuMemPoolSetAttributeShim;
    cudaFunctionMap["hipMemPoolGetAttribute"] = (void *)cuMemPoolGetAttributeShim;
    cudaFunctionMap["hipMemPoolSetAccess"] = (void *)cuMemPoolSetAccessShim;
    cudaFunctionMap["hipMemPoolGetAccess"] = (void *)cuMemPoolGetAccessShim;
    cudaFunctionMap["hipMemPoolCreate"] = (void *)cuMemPoolCreateShim;
    cudaFunctionMap["hipMemPoolDestroy"] = (void *)cuMemPoolDestroyShim;
    cudaFunctionMap["hipMemPoolExportToShareableHandle"] = (void *)cuMemPoolExportToShareableHandleShim;
    cudaFunctionMap["hipMemPoolImportFromShareableHandle"] = (void *)cuMemPoolImportFromShareableHandleShim;
    cudaFunctionMap["hipMemPoolExportPointer"] = (void *)cuMemPoolExportPointerShim;
    cudaFunctionMap["hipMemPoolImportPointer"] = (void *)cuMemPoolImportPointerShim;
    cudaFunctionMap["cuMemcpy"] = (void *)cuMemcpyShim;
    cudaFunctionMap["cuMemcpyAsync"] = (void *)cuMemcpyAsyncShim;
    cudaFunctionMap["cuMemcpyPeer"] = (void *)cuMemcpyPeerShim;
    cudaFunctionMap["cuMemcpyPeerAsync"] = (void *)cuMemcpyPeerAsyncShim;
    cudaFunctionMap["hipMemcpyHtoD"] = (void *)cuMemcpyHtoDShim;
    cudaFunctionMap["hipMemcpyHtoDAsync"] = (void *)cuMemcpyHtoDAsyncShim;
    cudaFunctionMap["hipMemcpyDtoH"] = (void *)cuMemcpyDtoHShim;
    cudaFunctionMap["hipMemcpyDtoHAsync"] = (void *)cuMemcpyDtoHAsyncShim;
    cudaFunctionMap["hipMemcpyDtoD"] = (void *)cuMemcpyDtoDShim;
    cudaFunctionMap["hipMemcpyDtoDAsync"] = (void *)cuMemcpyDtoDAsyncShim;
    cudaFunctionMap["hipDrvMemcpy2DUnaligned"] = (void *)cuMemcpy2DUnalignedShim;
    cudaFunctionMap["hipMemcpyParam2DAsync"] = (void *)cuMemcpy2DAsyncShim;
    cudaFunctionMap["hipDrvMemcpy3D"] = (void *)cuMemcpy3DShim;
    cudaFunctionMap["hipDrvMemcpy3DAsync"] = (void *)cuMemcpy3DAsyncShim;
    cudaFunctionMap["cuMemcpy3DPeer"] = (void *)cuMemcpy3DPeerShim;
    cudaFunctionMap["cuMemcpy3DPeerAsync"] = (void *)cuMemcpy3DPeerAsyncShim;
    cudaFunctionMap["hipMemsetD8"] = (void *)cuMemsetD8Shim;
    cudaFunctionMap["hipMemsetD8Async"] = (void *)cuMemsetD8AsyncShim;
    cudaFunctionMap["hipMemsetD2D8"] = (void *)cuMemsetD2D8Shim;
    cudaFunctionMap["hipMemsetD2D8Async"] = (void *)cuMemsetD2D8AsyncShim;
    cudaFunctionMap["cuFuncSetCacheConfig"] = (void *)cuFuncSetCacheConfigShim;
    cudaFunctionMap["cuFuncSetSharedMemConfig"] = (void *)cuFuncSetSharedMemConfigShim;
    cudaFunctionMap["hipFuncGetAttribute"] = (void *)cuFuncGetAttributeShim;
    cudaFunctionMap["cuFuncSetAttribute"] = (void *)cuFuncSetAttributeShim;
    cudaFunctionMap["hipArrayCreate"] = (void *)cuArrayCreateShim;
    cudaFunctionMap["hipArrayGetDescriptor"] = (void *)cuArrayGetDescriptorShim;
    cudaFunctionMap["cuArrayGetSparseProperties"] = (void *)cuArrayGetSparsePropertiesShim;
    cudaFunctionMap["cuArrayGetPlane"] = (void *)cuArrayGetPlaneShim;
    cudaFunctionMap["hipArray3DCreate"] = (void *)cuArray3DCreateShim;
    cudaFunctionMap["hipArray3DGetDescriptor"] = (void *)cuArray3DGetDescriptorShim;
    cudaFunctionMap["hipArrayDestroy"] = (void *)cuArrayDestroyShim;
    cudaFunctionMap["hipMipmappedArrayCreate"] = (void *)cuMipmappedArrayCreateShim;
    cudaFunctionMap["hipMipmappedArrayGetLevel"] = (void *)cuMipmappedArrayGetLevelShim;
    cudaFunctionMap["cuMipmappedArrayGetSparseProperties"] = (void *)cuMipmappedArrayGetSparsePropertiesShim;
    cudaFunctionMap["hipMipmappedArrayDestroy"] = (void *)cuMipmappedArrayDestroyShim;
    cudaFunctionMap["cuArrayGetMemoryRequirements"] = (void *)cuArrayGetMemoryRequirementsShim;
    cudaFunctionMap["cuMipmappedArrayGetMemoryRequirements"] = (void *)cuMipmappedArrayGetMemoryRequirementsShim;
    cudaFunctionMap["hipTexObjectCreate"] = (void *)cuTexObjectCreateShim;
    cudaFunctionMap["hipTexObjectDestroy"] = (void *)cuTexObjectDestroyShim;
    cudaFunctionMap["hipTexObjectGetResourceDesc"] = (void *)cuTexObjectGetResourceDescShim;
    cudaFunctionMap["hipTexObjectGetTextureDesc"] = (void *)cuTexObjectGetTextureDescShim;
    cudaFunctionMap["hipTexObjectGetResourceViewDesc"] = (void *)cuTexObjectGetResourceViewDescShim;
    cudaFunctionMap["cuSurfObjectCreate"] = (void *)cuSurfObjectCreateShim;
    cudaFunctionMap["cuSurfObjectDestroy"] = (void *)cuSurfObjectDestroyShim;
    cudaFunctionMap["cuSurfObjectGetResourceDesc"] = (void *)cuSurfObjectGetResourceDescShim;
    cudaFunctionMap["hipImportExternalMemory"] = (void *)cuImportExternalMemoryShim;
    cudaFunctionMap["hipExternalMemoryGetMappedBuffer"] = (void *)cuExternalMemoryGetMappedBufferShim;
    cudaFunctionMap["cuExternalMemoryGetMappedMipmappedArray"] = (void *)cuExternalMemoryGetMappedMipmappedArrayShim;
    cudaFunctionMap["hipDestroyExternalMemory"] = (void *)cuDestroyExternalMemoryShim;
    cudaFunctionMap["hipImportExternalSemaphore"] = (void *)cuImportExternalSemaphoreShim;
    cudaFunctionMap["hipSignalExternalSemaphoresAsync"] = (void *)cuSignalExternalSemaphoresAsyncShim;
    cudaFunctionMap["hipWaitExternalSemaphoresAsync"] = (void *)cuWaitExternalSemaphoresAsyncShim;
    cudaFunctionMap["hipDestroyExternalSemaphore"] = (void *)cuDestroyExternalSemaphoreShim;
    cudaFunctionMap["cuDeviceGetNvSciSyncAttributes"] = (void *)cuDeviceGetNvSciSyncAttributesShim;
    cudaFunctionMap["hipModuleLaunchKernel"] = (void *)cuLaunchKernelShim;
    cudaFunctionMap["hipModuleLaunchCooperativeKernel"] = (void *)cuLaunchCooperativeKernelShim;
    cudaFunctionMap["hipModuleLaunchCooperativeKernelMultiDevice"] = (void *)cuLaunchCooperativeKernelMultiDeviceShim;
    cudaFunctionMap["hipLaunchHostFunc"] = (void *)cuLaunchHostFuncShim;
    cudaFunctionMap["hipDrvLaunchKernelEx"] = (void *)cuLaunchKernelExShim;
    cudaFunctionMap["hipEventCreateWithFlags"] = (void *)cuEventCreateShim;
    cudaFunctionMap["hipEventRecord"] = (void *)cuEventRecordShim;
    cudaFunctionMap["hipEventRecordWithFlags"] = (void *)cuEventRecordWithFlagsShim;
    cudaFunctionMap["hipEventQuery"] = (void *)cuEventQueryShim;
    cudaFunctionMap["hipEventSynchronize"] = (void *)cuEventSynchronizeShim;
    cudaFunctionMap["hipEventDestroy"] = (void *)cuEventDestroyShim;
    cudaFunctionMap["hipEventElapsedTime"] = (void *)cuEventElapsedTimeShim;
    cudaFunctionMap["hipStreamWaitValue32"] = (void *)cuStreamWaitValue32Shim;
    cudaFunctionMap["hipStreamWriteValue32"] = (void *)cuStreamWriteValue32Shim;
    cudaFunctionMap["hipStreamWaitValue64"] = (void *)cuStreamWaitValue64Shim;
    cudaFunctionMap["hipStreamWriteValue64"] = (void *)cuStreamWriteValue64Shim;
    cudaFunctionMap["hipStreamBatchMemOp"] = (void *)cuStreamBatchMemOpShim;
    cudaFunctionMap["hipStreamCreateWithFlags"] = (void *)cuStreamCreateShim;
    cudaFunctionMap["hipStreamCreateWithPriority"] = (void *)cuStreamCreateWithPriorityShim;
    cudaFunctionMap["hipStreamGetPriority"] = (void *)cuStreamGetPriorityShim;
    cudaFunctionMap["hipStreamGetFlags"] = (void *)cuStreamGetFlagsShim;
    cudaFunctionMap["cuStreamGetCtx"] = (void *)cuStreamGetCtxShim;
    cudaFunctionMap["hipStreamGetId"] = (void *)cuStreamGetIdShim;
    cudaFunctionMap["hipStreamDestroy"] = (void *)cuStreamDestroyShim;
    cudaFunctionMap["hipStreamWaitEvent"] = (void *)cuStreamWaitEventShim;
    cudaFunctionMap["hipStreamAddCallback"] = (void *)cuStreamAddCallbackShim;
    cudaFunctionMap["hipStreamSynchronize"] = (void *)cuStreamSynchronizeShim;
    cudaFunctionMap["hipStreamQuery"] = (void *)cuStreamQueryShim;
    cudaFunctionMap["hipStreamAttachMemAsync"] = (void *)cuStreamAttachMemAsyncShim;
    cudaFunctionMap["hipStreamCopyAttributes"] = (void *)cuStreamCopyAttributesShim;
    cudaFunctionMap["hipStreamGetAttribute"] = (void *)cuStreamGetAttributeShim;
    cudaFunctionMap["hipStreamSetAttribute"] = (void *)cuStreamSetAttributeShim;
    cudaFunctionMap["hipDeviceCanAccessPeer"] = (void *)cuDeviceCanAccessPeerShim;
    cudaFunctionMap["hipCtxEnablePeerAccess"] = (void *)cuCtxEnablePeerAccessShim;
    cudaFunctionMap["hipCtxDisablePeerAccess"] = (void *)cuCtxDisablePeerAccessShim;
    cudaFunctionMap["hipIpcGetEventHandle"] = (void *)cuIpcGetEventHandleShim;
    cudaFunctionMap["hipIpcOpenEventHandle"] = (void *)cuIpcOpenEventHandleShim;
    cudaFunctionMap["hipIpcGetMemHandle"] = (void *)cuIpcGetMemHandleShim;
    cudaFunctionMap["hipIpcOpenMemHandle"] = (void *)cuIpcOpenMemHandleShim;
    cudaFunctionMap["hipIpcCloseMemHandle"] = (void *)cuIpcCloseMemHandleShim;
    cudaFunctionMap["cuGLCtxCreate"] = (void *)cuGLCtxCreateShim;
    cudaFunctionMap["cuGLInit"] = (void *)cuGLInitShim;
    cudaFunctionMap["hipGLGetDevices"] = (void *)cuGLGetDevicesShim;
    cudaFunctionMap["cuGLRegisterBufferObject"] = (void *)cuGLRegisterBufferObjectShim;
    cudaFunctionMap["cuGLMapBufferObject"] = (void *)cuGLMapBufferObjectShim;
    cudaFunctionMap["cuGLMapBufferObjectAsync"] = (void *)cuGLMapBufferObjectAsyncShim;
    cudaFunctionMap["cuGLUnmapBufferObject"] = (void *)cuGLUnmapBufferObjectShim;
    cudaFunctionMap["cuGLUnmapBufferObjectAsync"] = (void *)cuGLUnmapBufferObjectAsyncShim;
    cudaFunctionMap["cuGLUnregisterBufferObject"] = (void *)cuGLUnregisterBufferObjectShim;
    cudaFunctionMap["cuGLSetBufferObjectMapFlags"] = (void *)cuGLSetBufferObjectMapFlagsShim;
    cudaFunctionMap["hipGraphicsGLRegisterImage"] = (void *)cuGraphicsGLRegisterImageShim;
    cudaFunctionMap["hipGraphicsGLRegisterBuffer"] = (void *)cuGraphicsGLRegisterBufferShim;
    cudaFunctionMap["cuGraphicsEGLRegisterImage"] = (void *)cuGraphicsEGLRegisterImageShim;
    cudaFunctionMap["cuEGLStreamConsumerConnect"] = (void *)cuEGLStreamConsumerConnectShim;
    cudaFunctionMap["cuEGLStreamConsumerDisconnect"] = (void *)cuEGLStreamConsumerDisconnectShim;
    cudaFunctionMap["cuEGLStreamConsumerAcquireFrame"] = (void *)cuEGLStreamConsumerAcquireFrameShim;
    cudaFunctionMap["cuEGLStreamConsumerReleaseFrame"] = (void *)cuEGLStreamConsumerReleaseFrameShim;
    cudaFunctionMap["cuEGLStreamProducerConnect"] = (void *)cuEGLStreamProducerConnectShim;
    cudaFunctionMap["cuEGLStreamProducerDisconnect"] = (void *)cuEGLStreamProducerDisconnectShim;
    cudaFunctionMap["cuEGLStreamProducerPresentFrame"] = (void *)cuEGLStreamProducerPresentFrameShim;
    cudaFunctionMap["cuEGLStreamProducerReturnFrame"] = (void *)cuEGLStreamProducerReturnFrameShim;
    cudaFunctionMap["cuGraphicsResourceGetMappedEglFrame"] = (void *)cuGraphicsResourceGetMappedEglFrameShim;
    cudaFunctionMap["hipGraphicsUnregisterResource"] = (void *)cuGraphicsUnregisterResourceShim;
    cudaFunctionMap["hipGraphicsMapResources"] = (void *)cuGraphicsMapResourcesShim;
    cudaFunctionMap["hipGraphicsUnmapResources"] = (void *)cuGraphicsUnmapResourcesShim;
    cudaFunctionMap["cuGraphicsResourceSetMapFlags"] = (void *)cuGraphicsResourceSetMapFlagsShim;
    cudaFunctionMap["hipGraphicsSubResourceGetMappedArray"] = (void *)cuGraphicsSubResourceGetMappedArrayShim;
    cudaFunctionMap["cuGraphicsResourceGetMappedMipmappedArray"] = (void *)cuGraphicsResourceGetMappedMipmappedArrayShim;
    cudaFunctionMap["cuProfilerInitialize"] = (void *)cuProfilerInitializeShim;
    cudaFunctionMap["hipProfilerStart"] = (void *)cuProfilerStartShim;
    cudaFunctionMap["hipProfilerStop"] = (void *)cuProfilerStopShim;
    cudaFunctionMap["cuVDPAUGetDevice"] = (void *)cuVDPAUGetDeviceShim;
    cudaFunctionMap["cuVDPAUCtxCreate"] = (void *)cuVDPAUCtxCreateShim;
    cudaFunctionMap["cuGraphicsVDPAURegisterVideoSurface"] = (void *)cuGraphicsVDPAURegisterVideoSurfaceShim;
    cudaFunctionMap["cuGraphicsVDPAURegisterOutputSurface"] = (void *)cuGraphicsVDPAURegisterOutputSurfaceShim;
    cudaFunctionMap["cuGetExportTable"] = (void *)cuGetExportTableShim;
    cudaFunctionMap["hipModuleOccupancyMaxActiveBlocksPerMultiprocessorWithFlags"] = (void *)cuOccupancyMaxActiveBlocksPerMultiprocessorWithFlagsShim;
    cudaFunctionMap["hipOccupancyAvailableDynamicSMemPerBlock"] = (void *)cuOccupancyAvailableDynamicSMemPerBlockShim;
    cudaFunctionMap["cuOccupancyMaxPotentialClusterSize"] = (void *)cuOccupancyMaxPotentialClusterSizeShim;
    cudaFunctionMap["cuOccupancyMaxActiveClusters"] = (void *)cuOccupancyMaxActiveClustersShim;
    cudaFunctionMap["hipMemAdvise"] = (void *)cuMemAdviseShim;
    cudaFunctionMap["hipMemPrefetchAsync"] = (void *)cuMemPrefetchAsyncShim;
    cudaFunctionMap["hipMemRangeGetAttribute"] = (void *)cuMemRangeGetAttributeShim;
    cudaFunctionMap["hipMemRangeGetAttributes"] = (void *)cuMemRangeGetAttributesShim;
    cudaFunctionMap["hipDrvGetErrorString"] = (void *)cuGetErrorStringShim;
    cudaFunctionMap["hipDrvGetErrorName"] = (void *)cuGetErrorNameShim;
    cudaFunctionMap["hipGraphCreate"] = (void *)cuGraphCreateShim;
    cudaFunctionMap["hipGraphAddKernelNode"] = (void *)cuGraphAddKernelNodeShim;
    cudaFunctionMap["hipGraphKernelNodeGetParams"] = (void *)cuGraphKernelNodeGetParamsShim;
    cudaFunctionMap["hipGraphKernelNodeSetParams"] = (void *)cuGraphKernelNodeSetParamsShim;
    cudaFunctionMap["hipDrvGraphAddMemcpyNode"] = (void *)cuGraphAddMemcpyNodeShim;
    cudaFunctionMap["hipDrvGraphMemcpyNodeGetParams"] = (void *)cuGraphMemcpyNodeGetParamsShim;
    cudaFunctionMap["hipDrvGraphMemcpyNodeSetParams"] = (void *)cuGraphMemcpyNodeSetParamsShim;
    cudaFunctionMap["hipDrvGraphAddMemsetNode"] = (void *)cuGraphAddMemsetNodeShim;
    cudaFunctionMap["hipGraphMemsetNodeGetParams"] = (void *)cuGraphMemsetNodeGetParamsShim;
    cudaFunctionMap["hipGraphMemsetNodeSetParams"] = (void *)cuGraphMemsetNodeSetParamsShim;
    cudaFunctionMap["hipGraphAddHostNode"] = (void *)cuGraphAddHostNodeShim;
    cudaFunctionMap["hipGraphHostNodeGetParams"] = (void *)cuGraphHostNodeGetParamsShim;
    cudaFunctionMap["hipGraphHostNodeSetParams"] = (void *)cuGraphHostNodeSetParamsShim;
    cudaFunctionMap["hipGraphAddChildGraphNode"] = (void *)cuGraphAddChildGraphNodeShim;
    cudaFunctionMap["hipGraphChildGraphNodeGetGraph"] = (void *)cuGraphChildGraphNodeGetGraphShim;
    cudaFunctionMap["hipGraphAddEmptyNode"] = (void *)cuGraphAddEmptyNodeShim;
    cudaFunctionMap["hipGraphAddEventRecordNode"] = (void *)cuGraphAddEventRecordNodeShim;
    cudaFunctionMap["hipGraphEventRecordNodeGetEvent"] = (void *)cuGraphEventRecordNodeGetEventShim;
    cudaFunctionMap["hipGraphEventRecordNodeSetEvent"] = (void *)cuGraphEventRecordNodeSetEventShim;
    cudaFunctionMap["hipGraphAddEventWaitNode"] = (void *)cuGraphAddEventWaitNodeShim;
    cudaFunctionMap["hipGraphEventWaitNodeGetEvent"] = (void *)cuGraphEventWaitNodeGetEventShim;
    cudaFunctionMap["hipGraphEventWaitNodeSetEvent"] = (void *)cuGraphEventWaitNodeSetEventShim;
    cudaFunctionMap["hipGraphAddExternalSemaphoresSignalNode"] = (void *)cuGraphAddExternalSemaphoresSignalNodeShim;
    cudaFunctionMap["hipGraphExternalSemaphoresSignalNodeGetParams"] = (void *)cuGraphExternalSemaphoresSignalNodeGetParamsShim;
    cudaFunctionMap["hipGraphExternalSemaphoresSignalNodeSetParams"] = (void *)cuGraphExternalSemaphoresSignalNodeSetParamsShim;
    cudaFunctionMap["hipGraphAddExternalSemaphoresWaitNode"] = (void *)cuGraphAddExternalSemaphoresWaitNodeShim;
    cudaFunctionMap["hipGraphExternalSemaphoresWaitNodeGetParams"] = (void *)cuGraphExternalSemaphoresWaitNodeGetParamsShim;
    cudaFunctionMap["hipGraphExternalSemaphoresWaitNodeSetParams"] = (void *)cuGraphExternalSemaphoresWaitNodeSetParamsShim;
    cudaFunctionMap["hipGraphExecExternalSemaphoresSignalNodeSetParams"] = (void *)cuGraphExecExternalSemaphoresSignalNodeSetParamsShim;
    cudaFunctionMap["hipGraphExecExternalSemaphoresWaitNodeSetParams"] = (void *)cuGraphExecExternalSemaphoresWaitNodeSetParamsShim;
    cudaFunctionMap["hipGraphAddMemAllocNode"] = (void *)cuGraphAddMemAllocNodeShim;
    cudaFunctionMap["hipGraphMemAllocNodeGetParams"] = (void *)cuGraphMemAllocNodeGetParamsShim;
    cudaFunctionMap["hipDrvGraphAddMemFreeNode"] = (void *)cuGraphAddMemFreeNodeShim;
    cudaFunctionMap["hipGraphMemFreeNodeGetParams"] = (void *)cuGraphMemFreeNodeGetParamsShim;
    cudaFunctionMap["hipDeviceGraphMemTrim"] = (void *)cuDeviceGraphMemTrimShim;
    cudaFunctionMap["hipDeviceGetGraphMemAttribute"] = (void *)cuDeviceGetGraphMemAttributeShim;
    cudaFunctionMap["hipDeviceSetGraphMemAttribute"] = (void *)cuDeviceSetGraphMemAttributeShim;
    cudaFunctionMap["hipGraphClone"] = (void *)cuGraphCloneShim;
    cudaFunctionMap["hipGraphNodeFindInClone"] = (void *)cuGraphNodeFindInCloneShim;
    cudaFunctionMap["hipGraphNodeGetType"] = (void *)cuGraphNodeGetTypeShim;
    cudaFunctionMap["hipGraphGetNodes"] = (void *)cuGraphGetNodesShim;
    cudaFunctionMap["hipGraphGetRootNodes"] = (void *)cuGraphGetRootNodesShim;
    cudaFunctionMap["hipGraphGetEdges"] = (void *)cuGraphGetEdgesShim;
    cudaFunctionMap["hipGraphNodeGetDependencies"] = (void *)cuGraphNodeGetDependenciesShim;
    cudaFunctionMap["hipGraphNodeGetDependentNodes"] = (void *)cuGraphNodeGetDependentNodesShim;
    cudaFunctionMap["hipGraphAddDependencies"] = (void *)cuGraphAddDependenciesShim;
    cudaFunctionMap["hipGraphRemoveDependencies"] = (void *)cuGraphRemoveDependenciesShim;
    cudaFunctionMap["hipGraphDestroyNode"] = (void *)cuGraphDestroyNodeShim;
    cudaFunctionMap["hipGraphInstantiate"] = (void *)cuGraphInstantiateShim;
    cudaFunctionMap["hipGraphUpload"] = (void *)cuGraphUploadShim;
    cudaFunctionMap["hipGraphLaunch"] = (void *)cuGraphLaunchShim;
    cudaFunctionMap["hipGraphExecDestroy"] = (void *)cuGraphExecDestroyShim;
    cudaFunctionMap["hipGraphDestroy"] = (void *)cuGraphDestroyShim;
    cudaFunctionMap["hipStreamBeginCapture"] = (void *)cuStreamBeginCaptureShim;
    cudaFunctionMap["hipStreamEndCapture"] = (void *)cuStreamEndCaptureShim;
    cudaFunctionMap["hipStreamIsCapturing"] = (void *)cuStreamIsCapturingShim;
    cudaFunctionMap["hipStreamGetCaptureInfo"] = (void *)cuStreamGetCaptureInfoShim;
    cudaFunctionMap["hipStreamUpdateCaptureDependencies"] = (void *)cuStreamUpdateCaptureDependenciesShim;
    cudaFunctionMap["hipGraphExecKernelNodeSetParams"] = (void *)cuGraphExecKernelNodeSetParamsShim;
    cudaFunctionMap["hipDrvGraphExecMemcpyNodeSetParams"] = (void *)cuGraphExecMemcpyNodeSetParamsShim;
    cudaFunctionMap["hipDrvGraphExecMemsetNodeSetParams"] = (void *)cuGraphExecMemsetNodeSetParamsShim;
    cudaFunctionMap["hipGraphExecHostNodeSetParams"] = (void *)cuGraphExecHostNodeSetParamsShim;
    cudaFunctionMap["hipGraphExecChildGraphNodeSetParams"] = (void *)cuGraphExecChildGraphNodeSetParamsShim;
    cudaFunctionMap["hipGraphExecEventRecordNodeSetEvent"] = (void *)cuGraphExecEventRecordNodeSetEventShim;
    cudaFunctionMap["hipGraphExecEventWaitNodeSetEvent"] = (void *)cuGraphExecEventWaitNodeSetEventShim;
    cudaFunctionMap["hipThreadExchangeStreamCaptureMode"] = (void *)cuThreadExchangeStreamCaptureModeShim;
    cudaFunctionMap["hipGraphExecUpdate"] = (void *)cuGraphExecUpdateShim;
    cudaFunctionMap["hipGraphKernelNodeCopyAttributes"] = (void *)cuGraphKernelNodeCopyAttributesShim;
    cudaFunctionMap["hipGraphKernelNodeGetAttribute"] = (void *)cuGraphKernelNodeGetAttributeShim;
    cudaFunctionMap["hipGraphKernelNodeSetAttribute"] = (void *)cuGraphKernelNodeSetAttributeShim;
    cudaFunctionMap["hipGraphDebugDotPrint"] = (void *)cuGraphDebugDotPrintShim;
    cudaFunctionMap["hipUserObjectCreate"] = (void *)cuUserObjectCreateShim;
    cudaFunctionMap["hipUserObjectRetain"] = (void *)cuUserObjectRetainShim;
    cudaFunctionMap["hipUserObjectRelease"] = (void *)cuUserObjectReleaseShim;
    cudaFunctionMap["hipGraphRetainUserObject"] = (void *)cuGraphRetainUserObjectShim;
    cudaFunctionMap["hipGraphReleaseUserObject"] = (void *)cuGraphReleaseUserObjectShim;
    cudaFunctionMap["hipGraphNodeSetEnabled"] = (void *)cuGraphNodeSetEnabledShim;
    cudaFunctionMap["hipGraphNodeGetEnabled"] = (void *)cuGraphNodeGetEnabledShim;
    cudaFunctionMap["hipGraphInstantiateWithParams"] = (void *)cuGraphInstantiateWithParamsShim;
    cudaFunctionMap["hipGraphExecGetFlags"] = (void *)cuGraphExecGetFlagsShim;
    cudaFunctionMap["cuGraphInstantiateWithParams_ptsz"] = (void *)cuGraphInstantiateWithParams_ptszShim;
    cudaFunctionMap["hipGraphInstantiateWithFlags"] = (void *)cuGraphInstantiateWithFlagsShim;
    cudaFunctionMap["cuEGLStreamConsumerConnectWithFlags"] = (void *)cuEGLStreamConsumerConnectWithFlagsShim;
    cudaFunctionMap["hipGraphicsResourceGetMappedPointer"] = (void *)cuGraphicsResourceGetMappedPointerShim;
}

hipError_t cuGetProcAddress_v2_handler(const char *symbol, void **pfn, int cudaVersion, cuuint64_t flags, hipDriverProcAddressQueryResult *symbolStatus)
{
    initCudaFunctionMap();

    std::string symbolName(symbol);
    auto it = cudaFunctionMap.find(symbolName);
    if (it != cudaFunctionMap.end()) {
        *pfn = reinterpret_cast<void *>(it->second);
        std::cout << "Mapped symbol: " << symbolName << " to function: " << *pfn << std::endl;
    } else {
        std::cerr << "Function for symbol: " << symbolName << " not found!" << std::endl;
        *pfn = reinterpret_cast<void *>(noOpFunction); 
    }

    return hipSuccess;
}

hipError_t hipGetProcAddress(const char *symbol, void **pfn, int cudaVersion, cuuint64_t flags, hipDriverProcAddressQueryResult *symbolStatus) {
    open_rpc_client();

    std::string symbolName(symbol);
    auto it = cudaFunctionMap.find(symbolName);
    if (it != cudaFunctionMap.end()) {
        *pfn = reinterpret_cast<void *>(it->second);
        std::cout << "Mapped symbol: " << symbolName << " to function: " << *pfn << std::endl;
    } else {
        std::cerr << "Function for symbol: " << symbolName << " not found!" << std::endl;
        *pfn = reinterpret_cast<void *>(noOpFunction); 
    }
}

std::unordered_map<std::string, void*> functionMap;

void initializeFunctionMap()
{
    // simple cache check to make sure we only init handlers on the first run
    // attach all handlers to our function map
    functionMap["nvmlInitWithFlags"] = (void *)nvmlInitWithFlags;
    functionMap["nvmlInit_v2"] = (void *)nvmlInit_v2;
    functionMap["nvmlShutdown"] = (void *)nvmlShutdown;
    functionMap["nvmlSystemGetDriverVersion"] =
        (void *)nvmlSystemGetDriverVersion;
    functionMap["nvmlSystemGetHicVersion"] = (void *)nvmlSystemGetHicVersion;
    functionMap["nvmlSystemGetNVMLVersion"] = (void *)nvmlSystemGetNVMLVersion;
    functionMap["nvmlSystemGetProcessName"] = (void *)nvmlSystemGetProcessName;
    functionMap["nvmlSystemGetTopologyGpuSet"] =
        (void *)nvmlSystemGetTopologyGpuSet;
    functionMap["nvmlUnitGetCount"] = (void *)nvmlUnitGetCount;
    functionMap["nvmlUnitGetDevices"] = (void *)nvmlUnitGetDevices;
    functionMap["nvmlUnitGetFanSpeedInfo"] = (void *)nvmlUnitGetFanSpeedInfo;
    functionMap["nvmlUnitGetHandleByIndex"] = (void *)nvmlUnitGetHandleByIndex;
    functionMap["nvmlUnitGetLedState"] = (void *)nvmlUnitGetLedState;
    functionMap["nvmlUnitGetPsuInfo"] = (void *)nvmlUnitGetPsuInfo;
    functionMap["nvmlUnitGetTemperature"] = (void *)nvmlUnitGetTemperature;
    functionMap["nvmlUnitGetUnitInfo"] = (void *)nvmlUnitGetUnitInfo;

    // 4.16 Device Queries
    functionMap["nvmlDeviceGetClkMonStatus"] = (void *)nvmlDeviceGetClkMonStatus;
    functionMap["nvmlDeviceGetClock"] = (void *)nvmlDeviceGetClock;
    functionMap["nvmlDeviceGetClockInfo"] = (void *)nvmlDeviceGetClockInfo;
    functionMap["nvmlDeviceGetComputeMode"] = (void *)nvmlDeviceGetComputeMode;
    functionMap["nvmlDeviceGetCount_v2"] = (void *)nvmlDeviceGetCount_v2;
    functionMap["nvmlDeviceGetDisplayActive"] = (void *)nvmlDeviceGetDisplayActive;
    functionMap["nvmlDeviceGetDisplayMode"] = (void *)nvmlDeviceGetDisplayMode;
    functionMap["nvmlDeviceGetDriverModel_v2"] = (void *)nvmlDeviceGetDriverModel_v2;
    functionMap["nvmlDeviceGetDynamicPstatesInfo"] = (void *)nvmlDeviceGetDynamicPstatesInfo;
    functionMap["nvmlDeviceGetEccMode"] = (void *)nvmlDeviceGetEccMode;
    functionMap["nvmlDeviceGetEncoderCapacity"] = (void *)nvmlDeviceGetEncoderCapacity;
    functionMap["nvmlDeviceGetEncoderSessions"] = (void *)nvmlDeviceGetEncoderSessions;
    functionMap["nvmlDeviceGetEncoderStats"] = (void *)nvmlDeviceGetEncoderStats;
    functionMap["nvmlDeviceGetEncoderUtilization"] = (void *)nvmlDeviceGetEncoderUtilization;
    functionMap["nvmlDeviceGetEnforcedPowerLimit"] = (void *)nvmlDeviceGetEnforcedPowerLimit;
    functionMap["nvmlDeviceGetFanSpeed"] = (void *)nvmlDeviceGetFanSpeed;
    functionMap["nvmlDeviceGetHandleByIndex_v2"] =
        (void *)nvmlDeviceGetHandleByIndex_v2;
    functionMap["nvmlDeviceGetIndex"] = (void *)nvmlDeviceGetIndex;
    functionMap["nvmlDeviceGetMemoryInfo_v2"] =
        (void *)nvmlDeviceGetMemoryInfo_v2;
    functionMap["nvmlDeviceGetName"] = (void *)nvmlDeviceGetName;
    functionMap["nvmlDeviceGetPciInfo_v3"] = (void *)nvmlDeviceGetPciInfo_v3;
    functionMap["nvmlDeviceGetPcieSpeed"] = (void *)nvmlDeviceGetPcieSpeed;
    functionMap["nvmlDeviceGetPcieThroughput"] = (void *)nvmlDeviceGetPcieThroughput;
    functionMap["nvmlDeviceGetPerformanceState"] = (void *)nvmlDeviceGetPerformanceState;
    functionMap["nvmlDeviceGetPersistenceMode"] =
        (void *)nvmlDeviceGetPersistenceMode;
    functionMap["nvmlDeviceGetPowerSource"] = (void *)nvmlDeviceGetPowerSource;
    functionMap["nvmlDeviceGetPowerState"] = (void *)nvmlDeviceGetPowerState;
    functionMap["nvmlDeviceGetPowerUsage"] = (void *)nvmlDeviceGetPowerUsage;
    functionMap["nvmlDeviceGetSupportedPerformanceStates"] =
        (void *)nvmlDeviceGetSupportedPerformanceStates;
    functionMap["nvmlDeviceGetTargetFanSpeed"] = (void *)nvmlDeviceGetTargetFanSpeed;
    functionMap["nvmlDeviceGetTemperature"] = (void *)nvmlDeviceGetTemperature;
    functionMap["nvmlDeviceGetTemperatureThreshold"] = (void *)nvmlDeviceGetTemperatureThreshold;
    functionMap["nvmlDeviceGetThermalSettings"] = (void *)nvmlDeviceGetThermalSettings;
    functionMap["nvmlDeviceGetTopologyCommonAncestor"] = (void *)nvmlDeviceGetTopologyCommonAncestor;
    functionMap["nvmlDeviceGetTopologyNearestGpus"] = (void *)nvmlDeviceGetTopologyNearestGpus;
    functionMap["nvmlDeviceGetTotalEnergyConsumption"] = (void *)nvmlDeviceGetTotalEnergyConsumption;
    functionMap["nvmlDeviceGetUUID"] = (void *)nvmlDeviceGetUUID;
    functionMap["nvmlDeviceGetUtilizationRates"] = (void *)nvmlDeviceGetUtilizationRates;
    functionMap["nvmlDeviceValidateInforom"] = (void *)nvmlDeviceValidateInforom;

    // 4.17 Unit Commands
    functionMap["nvmlUnitSetLedState"] = (void *)nvmlUnitSetLedState;

    // 4.20 Event Handling Methods
    functionMap["nvmlDeviceGetSupportedEventTypes"] = (void *)nvmlDeviceGetSupportedEventTypes;
    functionMap["nvmlDeviceRegisterEvents"] = (void *)nvmlDeviceRegisterEvents;
    functionMap["nvmlEventSetCreate"] = (void *)nvmlEventSetCreate;
    functionMap["nvmlEventSetFree"] = (void *)nvmlEventSetFree;
    functionMap["nvmlEventSetWait_v2"] = (void *)nvmlEventSetWait_v2;

    // cuda
    functionMap["cuGetProcAddress_v2"] = (void *)cuGetProcAddress_v2_handler;

    functionMap["hipInit"] = (void *)hipInit;
    functionMap["hipGetProcAddress"] = (void *)hipGetProcAddress;
    functionMap["hipDriverGetVersion"] = (void *)cuDriverGetVersion_handler;
    functionMap["hipGetDeviceCount"] = (void *)hipGetDeviceCount;
    functionMap["hipDeviceGet"] = (void *)cuDeviceGetShim;
    functionMap["hipGetDeviceCount"] = (void *)cuDeviceGetCountShim;
    functionMap["hipDeviceGetName"] = (void *)cuDeviceGetNameShim;
    functionMap["hipDeviceTotalMem"] = (void *)cuDeviceTotalMemShim;
    functionMap["hipDeviceGetAttribute"] = (void *)cuDeviceGetAttributeShim;
    functionMap["hipDeviceGetP2PAttribute"] = (void *)cuDeviceGetP2PAttributeShim;
    functionMap["hipDeviceGetByPCIBusId"] = (void *)cuDeviceGetByPCIBusIdShim;
    functionMap["hipDeviceGetPCIBusId"] = (void *)cuDeviceGetPCIBusIdShim;
    functionMap["hipDeviceGetUuid"] = (void *)cuDeviceGetUuidShim;
    functionMap["cuDeviceGetTexture1DLinearMaxWidth"] = (void *)cuDeviceGetTexture1DLinearMaxWidthShim;
    functionMap["hipDeviceGetDefaultMemPool"] = (void *)cuDeviceGetDefaultMemPoolShim;
    functionMap["hipDeviceSetMemPool"] = (void *)cuDeviceSetMemPoolShim;
    functionMap["hipDeviceGetMemPool"] = (void *)cuDeviceGetMemPoolShim;
    functionMap["cuFlushGPUDirectRDMAWrites"] = (void *)cuFlushGPUDirectRDMAWritesShim;
    functionMap["hipDevicePrimaryCtxRetain"] = (void *)cuDevicePrimaryCtxRetainShim;
    functionMap["hipDevicePrimaryCtxRelease"] = (void *)cuDevicePrimaryCtxReleaseShim;
    functionMap["hipDevicePrimaryCtxSetFlags"] = (void *)cuDevicePrimaryCtxSetFlagsShim;
    functionMap["hipDevicePrimaryCtxGetState"] = (void *)cuDevicePrimaryCtxGetStateShim;
    functionMap["hipDevicePrimaryCtxReset"] = (void *)cuDevicePrimaryCtxResetShim;
    functionMap["hipCtxCreate"] = (void *)cuCtxCreateShim;
    functionMap["hipCtxGetFlags"] = (void *)cuCtxGetFlagsShim;
    functionMap["hipCtxSetCurrent"] = (void *)cuCtxSetCurrentShim;
    functionMap["hipCtxGetCurrent"] = (void *)cuCtxGetCurrentShim;
    functionMap["cuCtxDetach"] = (void *)cuCtxDetachShim;
    functionMap["hipCtxGetApiVersion"] = (void *)cuCtxGetApiVersionShim;
    functionMap["hipCtxGetDevice"] = (void *)cuCtxGetDeviceShim;
    functionMap["hipDeviceGetLimit"] = (void *)cuCtxGetLimitShim;
    functionMap["hipDeviceSetLimit"] = (void *)cuCtxSetLimitShim;
    functionMap["hipCtxGetCacheConfig"] = (void *)cuCtxGetCacheConfigShim;
    functionMap["hipCtxSetCacheConfig"] = (void *)cuCtxSetCacheConfigShim;
    functionMap["hipCtxGetSharedMemConfig"] = (void *)cuCtxGetSharedMemConfigShim;
    functionMap["hipDeviceGetStreamPriorityRange"] = (void *)cuCtxGetStreamPriorityRangeShim;
    functionMap["hipCtxSetSharedMemConfig"] = (void *)cuCtxSetSharedMemConfigShim;
    functionMap["hipCtxSynchronize"] = (void *)cuCtxSynchronizeShim;
    functionMap["cuCtxResetPersistingL2Cache"] = (void *)cuCtxResetPersistingL2CacheShim;
    functionMap["hipCtxPopCurrent"] = (void *)cuCtxPopCurrentShim;
    functionMap["hipCtxPushCurrent"] = (void *)cuCtxPushCurrentShim;
    functionMap["hipModuleLoad"] = (void *)cuModuleLoadShim;
    functionMap["hipModuleLoadData"] = (void *)cuModuleLoadDataShim;
    functionMap["hipModuleLoadFatBinary"] = (void *)cuModuleLoadFatBinaryShim;
    functionMap["hipModuleUnload"] = (void *)cuModuleUnloadShim;
    functionMap["hipModuleGetFunction"] = (void *)cuModuleGetFunctionShim;
    functionMap["hipModuleGetGlobal"] = (void *)cuModuleGetGlobalShim;
    functionMap["hipModuleGetTexRef"] = (void *)cuModuleGetTexRefShim;
    functionMap["cuModuleGetSurfRef"] = (void *)cuModuleGetSurfRefShim;
    functionMap["cuModuleGetLoadingMode"] = (void *)cuModuleGetLoadingModeShim;
    functionMap["hipLibraryLoadData"] = (void *)cuLibraryLoadDataShim;
    functionMap["hipLibraryLoadFromFile"] = (void *)cuLibraryLoadFromFileShim;
    functionMap["hipLibraryUnload"] = (void *)cuLibraryUnloadShim;
    functionMap["hipLibraryGetKernel"] = (void *)cuLibraryGetKernelShim;
    functionMap["cuLibraryGetModule"] = (void *)cuLibraryGetModuleShim;
    functionMap["cuKernelGetFunction"] = (void *)cuKernelGetFunctionShim;
    functionMap["cuLibraryGetGlobal"] = (void *)cuLibraryGetGlobalShim;
    functionMap["cuLibraryGetManaged"] = (void *)cuLibraryGetManagedShim;
    functionMap["cuKernelGetAttribute"] = (void *)cuKernelGetAttributeShim;
    functionMap["cuKernelSetAttribute"] = (void *)cuKernelSetAttributeShim;
    functionMap["cuKernelSetCacheConfig"] = (void *)cuKernelSetCacheConfigShim;
    functionMap["hiprtcLinkCreate"] = (void*)hiprtcLinkCreate;
    functionMap["hipGraphicsResourceGetMappedPointer"] = (void*)cuGraphicsResourceGetMappedPointerShim;
    functionMap["hipDrvGetErrorString"] = (void*)cuGetErrorStringShim;
    functionMap["hipDrvGetErrorName"] = (void*)cuGetErrorNameShim;
    functionMap["hiprtcLinkAddData"] = (void*)cuLinkAddData_v2Shim;
    functionMap["hiprtcLinkComplete"] = (void*)cuLinkCompleteShim;
    functionMap["hiprtcLinkDestroy"] = (void*)cuLinkDestroyShim;
    functionMap["hipModuleLoadDataEx"] = (void*)cuModuleLoadDataExShim;
    functionMap["hiprtcLinkAddFile"] = (void*)cuLinkAddFile_v2Shim;
    functionMap["cuFuncSetAttribute"] = (void*)cuFuncSetAttributeShim;
    functionMap["hipModuleLaunchKernel"] = (void*)cuLaunchKernelShim;
    functionMap["hipMemcpyDtoH"] = (void*)cuMemcpyDtoH_v2Shim;
    functionMap["hipStreamSynchronize"] = (void*)cuStreamSynchronizeShim;
    functionMap["hipModuleOccupancyMaxActiveBlocksPerMultiprocessor"] = (void*)cuOccupancyMaxActiveBlocksPerMultiprocessorShim;
    functionMap["hipDrvLaunchKernelEx"] = (void*)cuLaunchKernelExShim;
    functionMap["hipModuleGetGlobal"] = (void*)cuModuleGetGlobal_v2Shim;
    functionMap["hipFuncGetAttribute"] = (void*)cuFuncGetAttributeShim;
}

// Lookup function similar to dlsym
void *getFunctionByName(const char *name)
{
    auto it = functionMap.find(name);
    if (it != functionMap.end()) {
        std::cout << "Function found: " << name << it->second << std::endl;
        return it->second;
    }
        
    return nullptr;
}

void *dlsym(void *handle, const char *name) __THROW
{
    initializeFunctionMap();  // Initialize the function map

    void *func = getFunctionByName(name);  // Lookup function by name

    if (func != nullptr) {
        std::cout << "[dlsym] Function address from cudaFunctionMap: " << func << std::endl;
        return func;
    }

    // Real dlsym lookup
    static void *(*real_dlsym)(void *, const char *) = NULL;
    if (real_dlsym == NULL) {
        real_dlsym = (void *(*)(void *, const char *))dlvsym(RTLD_NEXT, "dlsym", "GLIBC_2.2.5");
    }

    std::cout << "[dlsym] Falling back to real_dlsym for name: " << name << std::endl;
    return real_dlsym(handle, name);
}
